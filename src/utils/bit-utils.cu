#include "hip/hip_runtime.h"
//#pragma once
// error: #pragma once in main file
#ifndef TURBONERF_BIT_UTILS_CU
#define TURBONERF_BIT_UTILS_CU

#include "bit-utils.cuh"

TURBO_NAMESPACE_BEGIN

// counts the bits whose value is 1, given some buffer
// written by James Perlman, with a tiny bit of help from GPT-4!

__global__ void get_1s_per_uint32(
    const size_t n_elements,
    const uint32_t* __restrict__ buffer,
    uint8_t* __restrict__ counts
) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_elements) return;

    const uint32_t value = buffer[idx];
    uint8_t count = static_cast<uint8_t>(__popc(value));
    counts[idx] = count;
}

TURBO_NAMESPACE_END

#endif
