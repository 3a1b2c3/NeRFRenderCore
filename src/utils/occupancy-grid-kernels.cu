#include "hip/hip_runtime.h"
#include "occupancy-grid-kernels.cuh"

/**
 * This entire file is a direct implementation of page 15, "Updating the occupancy grids" of this paper:
 * 
 * Müller, Thomas, et al. "Instant neural graphics primitives with a multiresolution hash encoding."
 * *ACM Trans. Graph.*, 41(4), 102:1-102:15 - https://doi.org/10.1145/3528223.3530127  
 */

using namespace tcnn;

NRC_NAMESPACE_BEGIN

// occupancy cell values first get decayed by a factor (default 0.95) every update
__global__ void decay_occupancy_grid_values_kernel(
    const uint32_t n_cells_per_level,
    const uint32_t n_levels,
    const float factor,
    float* __restrict__ grid_density
) {
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_cells_per_level) {
        return;
    }

    float* d = grid_density + idx;

    #pragma unroll
    for (int i = 0; i < n_levels; ++i) {
        *d *= factor;
        d += n_cells_per_level;
    }
}

// generate points in the grid for sampling the sigma network
// the network_pos values are in the range [0, 1]
__global__ void generate_grid_cell_network_sample_points_kernel(
    const uint32_t n_cells,
    const uint32_t batch_size,
    const uint32_t start_idx,
    const CascadedOccupancyGrid* __restrict__ grid,
    const int level,
    const float inv_aabb_size,
    const float* __restrict__ random_float,
    float* __restrict__ sample_pos
) {
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) {
        return;
    }
    
    const uint32_t idx = i + start_idx;

    const float level_size = grid->get_level_size(level);
    const float voxel_size = level_size * grid->inv_resolution_f;
    
    const uint32_t i_offset_0 = i;
    const uint32_t i_offset_1 = i_offset_0 + batch_size;
    const uint32_t i_offset_2 = i_offset_1 + batch_size;

    // get xyz positions of the grid cell according to the morton code index
    float vx, vy, vz;
    grid->get_voxel_xyz_from_morton_index(idx, vx, vy, vz);

    // origin of the grid cell (same value for all 3 axes)
    // this also centers xyz in the grid cell
    const float o = -0.5f * level_size;

    // set each dimension of sample_pos to the corner of the grid cell + a random offset
    // this x,y,z is in world coordinates
    const float x = o + ((float)vx + random_float[i_offset_0]) * voxel_size;
    const float y = o + ((float)vy + random_float[i_offset_1]) * voxel_size;
    const float z = o + ((float)vz + random_float[i_offset_2]) * voxel_size;

    // Normalize the sample position to the range [0, 1] (for the network)
    sample_pos[i_offset_0] = x * inv_aabb_size + 0.5f;
    sample_pos[i_offset_1] = y * inv_aabb_size + 0.5f;
    sample_pos[i_offset_2] = z * inv_aabb_size + 0.5f;

}

// occupancy cell values are updated to the maximum of the current value and a newly sampled density value
__global__ void update_occupancy_with_density_kernel(
    const uint32_t n_samples,
    const uint32_t start_idx,
    const CascadedOccupancyGrid* __restrict__ grid,
    const uint32_t level,
    const float selection_threshold,
    const float* __restrict__ random_float,
    const tcnn::network_precision_t* __restrict__ network_density
) {
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_samples) {
        return;
    }

    const uint32_t idx = i + start_idx;

    // (selection_threshold * 100)% of cells are sampled randomly, and the rest are sampled based on the current occupancy
    if (selection_threshold < random_float[idx] && !grid->is_occupied_at(level, idx)) {
        return;
    }

    float* grid_density = grid->get_density() + level * grid->volume_i + idx;
    float new_density = fmaxf(*grid_density, (float)network_density[i]);

    // if grid density is NaN, reset it to zero
    // if (isnan(new_density) || isinf(new_density)) {
    //     new_density = 0.0f;
    // }
    
    *grid_density = new_density;
}

// occupancy bits are updated by thresholding each cell's density, default = 0.01 * 1024 / sqrt(3)
__global__ void update_occupancy_grid_bits_kernel(
    const uint32_t n_cells_per_level,
    const int n_levels,
    const float threshold,
    const float* __restrict__ grid_density,
    uint8_t* __restrict__ grid_bits
) {
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_cells_per_level) {
        return;
    }

    uint8_t cell_bits = grid_bits[idx];

    #pragma unroll
    for (int level = 0; level < n_levels; ++level) {
        const uint32_t density_idx = level * n_cells_per_level + idx;

        // get "is threshold exceeded?" as a bit
        uint8_t b = grid_density[density_idx] > threshold ? 1 : 0;

        // thank you https://stackoverflow.com/a/28360719/892990
        // This just sets the bit at the correct position to the value of b
        cell_bits = (cell_bits & (~((uint8_t)1 << level))) | (b << level);
    }

    grid_bits[idx] = cell_bits;
}

NRC_NAMESPACE_END
