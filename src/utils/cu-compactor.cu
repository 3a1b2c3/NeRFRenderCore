#include "hip/hip_runtime.h"

/*
 * cuCompactor.cu
 *
 *  Created on: 21/mag/2015
 *      Author: knotman
 *
 * Modified on January 7, 2023 by James Perlman
 * Please see LICENSES/knotman90_cuStreamComp.md for license information
 * 
 * original code from https://github.com/knotman90/cuStreamComp
 */

#pragma once

#include <thrust/scan.h>
#include <thrust/device_vector.h>

#include "cu-compactor.cuh"

NRC_NAMESPACE_BEGIN

#define warpSize (32)
#define FULL_MASK 0xffffffff

__host__ __device__ int divup(int x, int y)
{
    return x / y + (x % y ? 1 : 0);
}

__device__ __inline__ int pow2i(int e)
{
    return 1 << e;
}

// predicate is an array of bools
__global__ void computeBlockCounts(
    int n_elements,
    const bool* __restrict__ d_predicate,
    int *d_BlockCounts
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx >= n_elements) return;

    int pred = d_predicate[idx];
    int BC = __syncthreads_count(pred);

    if (threadIdx.x == 0)
    {
        d_BlockCounts[blockIdx.x] = BC; // BC will contain the number of valid elements in all threads of this thread block
    }
}

__global__ void compactK(
    const int n_elements,
    const bool* __restrict__ d_predicate,
    const int* __restrict__ d_BlocksOffset,
    int* __restrict__ d_output
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ int warpTotals[];

    if (idx >= n_elements) return;

    int pred = d_predicate[idx];
    int w_i = threadIdx.x / warpSize; // warp index
    int w_l = idx % warpSize;         // thread index within a warp

    // compute exclusive prefix sum based on predicate validity to get output offset for thread in warp
    int t_m = FULL_MASK >> (warpSize - w_l); // thread mask
    int b = __ballot_sync(FULL_MASK, pred) & t_m;
    int t_u = __popc(b); // popc count the number of bit one. simply count the number predicated true BEFORE MY INDEX

    // last thread in warp computes total valid counts for the warp
    if (w_l == warpSize - 1) {
        warpTotals[w_i] = t_u + pred;
    }

    // need all warps in thread block to fill in warpTotals before proceeding
    __syncthreads();

    // first numWarps threads in first warp compute exclusive prefix sum to get output offset for each warp in thread block
    int numWarps = blockDim.x / warpSize;
    unsigned int numWarpsMask = FULL_MASK >> (warpSize - numWarps);
    if (w_i == 0 && w_l < numWarps)
    {
        int w_i_u = 0;
        for (int j = 0; j <= 5; j++)
        { // must include j=5 in loop in case any elements of warpTotals are identically equal to 32
            int b_j = __ballot_sync(numWarpsMask, warpTotals[w_l] & pow2i(j));
            w_i_u += (__popc(b_j & t_m)) << j;
            // printf("indice %i t_m=%i,j=%i,b_j=%i,w_i_u=%i\n",w_l,t_m,j,b_j,w_i_u);
        }
        warpTotals[w_l] = w_i_u;
    }

    // need all warps in thread block to wait until prefix sum is calculated in warpTotals
    __syncthreads();

    // if valid element, place the element in proper destination address based on thread offset in warp, warp offset in block, and block offset in grid
    if (pred) {
        d_output[t_u + warpTotals[w_i] + d_BlocksOffset[blockIdx.x]] = idx;
    }
}

template <class T>
__global__ void printArray_GPU(T *hd_data, int size, int newline)
{
    int w = 0;
    for (int i = 0; i < size; i++)
    {
        if (i % newline == 0)
        {
            printf("\n%i -> ", w);
            w++;
        }
        printf("%i ", hd_data[i]);
    }
    printf("\n");
}

int generate_compaction_indices(
    const hipStream_t& stream,
    const int n_elements,
    const int blockSize,
    const bool* d_predicate,
    int* d_output
) {
    int numBlocks = divup(n_elements, blockSize);
    int *d_BlocksCountAndOffset;

    // TODO: these cudaMallocs can be moved out to avoid unnecessary allocations
    CUDA_CHECK_THROW(hipMallocAsync(&d_BlocksCountAndOffset, 2 * numBlocks * sizeof(int), stream));

    thrust::device_ptr<int> bCount_ptr(d_BlocksCountAndOffset + 0 * numBlocks);
    thrust::device_ptr<int> bOffset_ptr(d_BlocksCountAndOffset + 1 * numBlocks);

    // phase 1: count number of valid elements in each thread block
    computeBlockCounts<<<numBlocks, blockSize, 0, stream>>>(
        n_elements,
        d_predicate,
        bCount_ptr.get()
    );

    // phase 2: compute exclusive prefix sum of valid block counts to get output offset for each thread block in grid
    thrust::exclusive_scan(
        thrust::cuda::par_nosync.on(stream),
        bCount_ptr,
        bCount_ptr + numBlocks,
        bOffset_ptr
    );

    // phase 3: compute output offset for each thread in warp and each warp in thread block, then output valid elements
    compactK<<<numBlocks, blockSize, sizeof(int) * (blockSize / warpSize), stream>>>(
        n_elements,
        d_predicate,
        bOffset_ptr.get(),
        d_output
    );

    // determine number of elements in the compacted list

    // copy last element of thrustPtr_bOffset to host
    int bCount, bOffset;
    CUDA_CHECK_THROW(hipMemcpyAsync(&bOffset, bOffset_ptr.get() + numBlocks - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_THROW(hipMemcpyAsync(&bCount, bCount_ptr.get() + numBlocks - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_THROW(hipStreamSynchronize(stream))
    // compact_length = thrustPrt_bOffset[numBlocks - 1] + thrustPrt_bCount[numBlocks - 1];
    hipFree(d_BlocksCountAndOffset);

    return bOffset + bCount;
}

NRC_NAMESPACE_END
