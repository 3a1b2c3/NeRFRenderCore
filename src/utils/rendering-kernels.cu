#include "hip/hip_runtime.h"

#include <>
#include <crt/hip/device_functions.h>

#include "rendering-kernels.cuh"
#include "../models/bounding-box.cuh"
#include "../models/camera.cuh"
#include "../models/cascaded-occupancy-grid.cuh"

using namespace tcnn;

NRC_NAMESPACE_BEGIN

// TODO: move this into a Camera utility kernel file
// init_rays CUDA kernel
__global__ void generate_rays_pinhole_kernel(
	const uint32_t n_rays,
	const uint32_t batch_size,
	const BoundingBox* __restrict__ bbox,
	const Camera* __restrict__ cam,
	float* __restrict__ ray_ori,
	float* __restrict__ ray_dir,
	float* __restrict__ ray_idir,
	float* __restrict__ ray_t,
    uint32_t* __restrict__ ray_idx,
	bool* __restrict__ ray_alive,
	const uint32_t start_idx
) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= n_rays) {
		return;
	}

	uint32_t idx = start_idx + i;
	
	uint32_t x = idx % cam->pixel_dims.x;
	uint32_t y = idx / cam->pixel_dims.x;

	Ray local_ray = cam->local_ray_at_pixel_xy(x, y);

    float3 global_origin = cam->transform * local_ray.o;
	float3 global_direction = cam->transform * local_ray.d - cam->transform.get_translation();

	// normalize ray directions
	const float n = rnorm3df(global_direction.x, global_direction.y, global_direction.z);

	const float dir_x = n * global_direction.x;
	const float dir_y = n * global_direction.y;
	const float dir_z = n * global_direction.z;

	const float idir_x = 1.0f / dir_x;
	const float idir_y = 1.0f / dir_y;
	const float idir_z = 1.0f / dir_z;

    // save data to buffers
	uint32_t i_offset_0 = i;
	uint32_t i_offset_1 = i_offset_0 + batch_size;
	uint32_t i_offset_2 = i_offset_1 + batch_size;

	ray_ori[i_offset_0] = global_origin.x;
	ray_ori[i_offset_1] = global_origin.y;
	ray_ori[i_offset_2] = global_origin.z;

	ray_dir[i_offset_0] = dir_x;
	ray_dir[i_offset_1] = dir_y;
	ray_dir[i_offset_2] = dir_z;

	ray_idir[i_offset_0] = idir_x;
	ray_idir[i_offset_1] = idir_y;
	ray_idir[i_offset_2] = idir_z;

	float t;
	const bool intersects_bbox = bbox->get_ray_t_intersection(
		global_origin.x, global_origin.y, global_origin.z,
		dir_x, dir_y, dir_z,
		idir_x, idir_y, idir_z,
		t
	);

	ray_t[i] = intersects_bbox ? fmaxf(0.0f, t + 1e-5f) : 0.0f;

	ray_alive[i] = intersects_bbox;

    ray_idx[i] = idx;
}

__global__ void march_rays_and_generate_network_inputs_kernel(
    const uint32_t n_rays,
	const uint32_t batch_size,
	const uint32_t network_stride,
	const CascadedOccupancyGrid* occ_grid,
	const BoundingBox* bbox,
	const float inv_aabb_size,
	const float dt_min,
	const float dt_max,
	const float cone_angle,
	
	// input buffers (read-only)
	const float* __restrict__ ray_ori,
	const float* __restrict__ ray_dir,
	const float* __restrict__ ray_idir,

    // dual-use buffers (read/write)
    bool* __restrict__ ray_alive,
    bool* __restrict__ ray_active,
    float* __restrict__ ray_t,

	// output buffers (write-only)
	float* __restrict__ network_pos,
	float* __restrict__ network_dir,
	float* __restrict__ network_dt
) {
	// get thread index
	const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

	// check if thread is out of bounds
	if (i >= n_rays) return;

    // check if ray has terminated or is currently inactive
    if (!ray_alive[i] || !ray_active[i]) return;

	// References to input buffers
	const uint32_t i_offset_0 = i;
	const uint32_t i_offset_1 = i_offset_0 + batch_size;
	const uint32_t i_offset_2 = i_offset_1 + batch_size;

	const uint32_t net_offset_0 = i;
	const uint32_t net_offset_1 = net_offset_0 + network_stride;
	const uint32_t net_offset_2 = net_offset_1 + network_stride;

	const float o_x = ray_ori[i_offset_0];
	const float o_y = ray_ori[i_offset_1];
	const float o_z = ray_ori[i_offset_2];

	const float d_x = ray_dir[i_offset_0];
	const float d_y = ray_dir[i_offset_1];
	const float d_z = ray_dir[i_offset_2];
	
	const float id_x = ray_idir[i_offset_0];
	const float id_y = ray_idir[i_offset_1];
	const float id_z = ray_idir[i_offset_2];

	// Perform raymarching

	float t = ray_t[i];

	while (true) {
		const float t0 = t;
		const float dt = occ_grid->get_dt(t, cone_angle, dt_min, dt_max);
		t += dt;
		const float tmid = 0.5f * (t0 + t);

		const float x = o_x + tmid * d_x;
		const float y = o_y + tmid * d_y;
		const float z = o_z + tmid * d_z;

		if (!bbox->contains(x, y, z)) {
			ray_alive[i] = false;
			break;
		}

		const int grid_level = occ_grid->get_grid_level_at(x, y, z, dt);

		if (occ_grid->is_occupied_at(grid_level, x, y, z)) {
			ray_t[i] = tmid;

			network_pos[net_offset_0] = x * inv_aabb_size + 0.5f;
			network_pos[net_offset_1] = y * inv_aabb_size + 0.5f;
			network_pos[net_offset_2] = z * inv_aabb_size + 0.5f;
			
			network_dir[net_offset_0] = (d_x + 1.0f) * 0.5f;
			network_dir[net_offset_1] = (d_y + 1.0f) * 0.5f;
			network_dir[net_offset_2] = (d_z + 1.0f) * 0.5f;

			network_dt[i] = dt * inv_aabb_size;

            // for now, we only march samples once.
            break;
			
			// t += dt;

		} else {
			// otherwise we need to find the next occupied cell
			t += occ_grid->get_dt_to_next_voxel(
				x, y, z,
				d_x, d_y, d_z,
				id_x, id_y, id_z,
				dt_min,
				grid_level
			);
		}
	}
}

// ray compaction
__global__ void compact_rays_kernel(
    const int n_compacted_elements,
	const int batch_size,
    const int* __restrict__ indices,

	// input buffers (read-only)
	const uint32_t* __restrict__ in_idx, // this is the ray-pixel index
	const bool* __restrict__ in_active,
	const float* __restrict__ in_t,
	const float* __restrict__ in_origin,
	const float* __restrict__ in_dir,
	const float* __restrict__ in_idir,
	const float* __restrict__ in_sigma,

	// compacted output buffers (write-only)
	uint32_t* __restrict__ out_idx,
	bool* __restrict__ out_active,
	float* __restrict__ out_t,
	float* __restrict__ out_origin,
	float* __restrict__ out_dir,
	float* __restrict__ out_idir,
	float* __restrict__ out_sigma
) {
    // compacted index is the index to write to
    const int c_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (c_idx >= n_compacted_elements) return;

	// expanded index is the index to read from
	const int e_idx = indices[c_idx];
	
	// 1-component buffers
	out_idx[c_idx]		= in_idx[e_idx];
	out_active[c_idx]	= in_active[e_idx];
	out_t[c_idx]		= in_t[e_idx];
	out_sigma[c_idx]	= in_sigma[e_idx];

	// local references to pointer offsets
	const int c_offset_0 = c_idx;
	const int c_offset_1 = c_offset_0 + batch_size;
	const int c_offset_2 = c_offset_1 + batch_size;

	const int e_offset_0 = e_idx;
	const int e_offset_1 = e_offset_0 + batch_size;
	const int e_offset_2 = e_offset_1 + batch_size;

	// 3-component buffers
	out_origin[c_offset_0] = in_origin[e_offset_0];
	out_origin[c_offset_1] = in_origin[e_offset_1];
	out_origin[c_offset_2] = in_origin[e_offset_2];
	
	out_dir[c_offset_0] = in_dir[e_offset_0];
	out_dir[c_offset_1] = in_dir[e_offset_1];
	out_dir[c_offset_2] = in_dir[e_offset_2];

	out_idir[c_offset_0] = in_idir[e_offset_0];
	out_idir[c_offset_1] = in_idir[e_offset_1];
	out_idir[c_offset_2] = in_idir[e_offset_2];

}

// alpha compositing kernel, composites the latest samples into the output image
__global__ void composite_samples_kernel(
    const uint32_t n_samples,
	const uint32_t network_stride,
	const uint32_t output_stride,
    
    // read-only
    const network_precision_t* __restrict__ network_output,
    const float* __restrict__ sample_dt,
    const uint32_t* __restrict__ sample_idx,
	const bool* __restrict__ ray_active,

    // read/write
    bool* __restrict__ ray_alive,
	float* __restrict__ ray_sigma,
    float* __restrict__ output_rgba
) {
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_samples) return;

    // check if ray has terminated or is currently inactive
    if (!ray_alive[i] || !ray_active[i]) return;

    // grab local references to global memory

    // sample colors
    const uint32_t net_offset_0 = i;
    const uint32_t net_offset_1 = net_offset_0 + network_stride;
    const uint32_t net_offset_2 = net_offset_1 + network_stride;
	const uint32_t net_offset_3 = net_offset_2 + network_stride;

	const float s_r = (float)network_output[net_offset_0];
	const float s_g = (float)network_output[net_offset_1];
	const float s_b = (float)network_output[net_offset_2];
	const float s_s = (float)network_output[net_offset_3];

	// sample sigma
	const float sigma_dt = s_s * sample_dt[i];

    // sample alpha
    const float s_a = 1.0f - __expf(-sigma_dt);

	// ray transmittance
	const float r_t = __expf(-ray_sigma[i]);

	if (r_t <= 1e-4f) {
		ray_alive[i] = false;
		return;
	}

	// sample weight
	const float s_w = s_a * r_t;

	// sigma cumulative sum
	ray_sigma[i] += sigma_dt;

    // pixel colors
    const uint32_t idx_offset_0 = sample_idx[i];
    const uint32_t idx_offset_1 = idx_offset_0 + output_stride;
    const uint32_t idx_offset_2 = idx_offset_1 + output_stride;
    const uint32_t idx_offset_3 = idx_offset_2 + output_stride;

	// composite the same way we do accumulation during training
	output_rgba[idx_offset_0] += s_w * s_r;
	output_rgba[idx_offset_1] += s_w * s_g;
	output_rgba[idx_offset_2] += s_w * s_b;
	output_rgba[idx_offset_3] += s_w;

	// terminate ray if alpha >= 1.0
	const float out_a = output_rgba[idx_offset_3];

	if (out_a >= 1.0f) {
		ray_alive[i] = false;
	
		output_rgba[idx_offset_0] /= out_a;
		output_rgba[idx_offset_1] /= out_a;
		output_rgba[idx_offset_2] /= out_a;
		output_rgba[idx_offset_3] = 1.0f;
	}
}

NRC_NAMESPACE_END
