#include "render-buffer.cuh"
#include "../utils/gpu-image.cuh"

using namespace nrc;

void RenderBuffer::clear(const hipStream_t& stream) {
    CUDA_CHECK_THROW(
        hipMemsetAsync(rgba, 0, width * height * 4 * sizeof(float), stream)
    );
}

void RenderBuffer::save_image(const std::string& filename, const hipStream_t& stream) {
    CUDA_CHECK_THROW(hipStreamSynchronize(stream));
    save_buffer_to_image(stream, filename, rgba, width, height, 4, stride);
}

std::vector<float> RenderBuffer::get_image(const hipStream_t& stream) {
    return save_buffer_to_memory(stream, rgba, width, height, 4, stride);
}
