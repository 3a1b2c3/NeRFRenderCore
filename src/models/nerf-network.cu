#include "hip/hip_runtime.h"
// This code was adapted from nerfstudio (Copyright 2022 The Nerfstudio Team. All rights reserved.)
// https://github.com/nerfstudio-project/nerfstudio/blob/main/nerfstudio/fields/instant_ngp_field.py
// Please see LICENSES/nerfstudio-project_nerfstudio.md for license details.

#include <json/json.hpp>
#include <math.h>
#include <tiny-cuda-nn/common.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "../utils/gpu-image.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/training-network-kernels.cuh"
#include "nerf-network.h"

using namespace tcnn;
using namespace nrc;
using json = nlohmann::json;


#if TCNN_HALF_PRECISION
    constexpr float LOSS_SCALE = 128.0f;
#else
    constexpr float LOSS_SCALE = 1.0f;
#endif


// Constructor

NerfNetwork::NerfNetwork(const float& aabb_size) {
	this->aabb_size = aabb_size;

	// TODO: set this properly based on the aabb
	double per_level_scale = 1.4472692012786865;

	// Create the Direction Encoding
	json direction_encoding_config = {
		{"otype", "SphericalHarmonics"},
		{"degree", 4},
	};

	direction_encoding.reset(
		create_encoding<network_precision_t>(3, direction_encoding_config)
	);

	// Create the Density MLP
	
	json density_encoding_config = {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", per_level_scale},
	};

	json density_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 1},
	};

	density_network.reset(
		new NetworkWithInputEncoding<network_precision_t>(
			3,	// input dims
			16, // output dims
			density_encoding_config,
			density_network_config
		)
	);

	// Create the Color MLP

	uint32_t color_network_in_dim = direction_encoding->padded_output_width() + density_network->padded_output_width();

	const json color_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
		{"n_input_dims", color_network_in_dim},
		{"n_output_dims", 3},
	};

	color_network.reset(
		create_network<network_precision_t>(color_network_config)
	);
}

// initialize params and gradients for the networks (I have no idea if this is correct)
void NerfNetwork::prepare_for_training(const hipStream_t& stream) {

	size_t rng_seed = 72791;
	pcg32 rng(rng_seed);

	// initialize params
	params_workspace.enlarge(
		stream,
		density_network->n_params(),
		color_network->n_params()
	);
	
	density_network->initialize_params(rng, params_workspace.density_network_params_fp);
	color_network->initialize_params(rng, params_workspace.color_network_params_fp);

	// initialize_params only initializes full precision params, need to copy to half precision

	copy_and_cast<network_precision_t, float>(
		stream,
		density_network->n_params(),
		params_workspace.density_network_params_hp,
		params_workspace.density_network_params_fp
	);

	copy_and_cast<network_precision_t, float>(
		stream,
		color_network->n_params(),
		params_workspace.color_network_params_hp,
		params_workspace.color_network_params_fp
	);

	// assign params pointers

	density_network->set_params(
		params_workspace.density_network_params_hp,
		params_workspace.density_network_params_hp,
		params_workspace.density_network_gradients_hp
	);

	color_network->set_params(
		params_workspace.color_network_params_hp,
		params_workspace.color_network_params_hp,
		params_workspace.color_network_gradients_hp
	);

	// initialize optimizers
	
	json optimizer_config = {
		{"otype", "Adam"},
		{"learning_rate", 1e-2},
		{"epsilon", 1e-15},
		{"l2_reg", 1e-6},
	};

	density_optimizer.reset(
		create_optimizer<network_precision_t>(optimizer_config)
	);

	density_optimizer->allocate(density_network);

	color_optimizer.reset(
		create_optimizer<network_precision_t>(optimizer_config)
	);
	
	color_optimizer->allocate(color_network);

	// flag for training enabled
	can_train = true;
}

void NerfNetwork::train(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	uint32_t* ray_steps,
	uint32_t* ray_steps_cum,
	float* pos_batch,
	float* dir_batch,
	float* dt_batch,
	float* target_rgba,
	network_precision_t* network_sigma,
	network_precision_t* network_color
) {
	
	enlarge_workspace_if_needed(stream, batch_size);

	// Forward
	auto fwd_ctx = forward(
		stream,
		batch_size,
		pos_batch,
		dir_batch,
		network_sigma,
		network_color
	);

	// Loss
	float mse_loss = calculate_loss(
		stream,
		batch_size,
		n_rays,
		n_samples,
		ray_steps,
		ray_steps_cum,
		dt_batch,
		target_rgba,
		network_sigma,
		network_color
	);

	printf("Loss: %f / # Rays: %lu\n", mse_loss, n_rays);

	// Backward
	backward(stream, fwd_ctx, batch_size, pos_batch, dir_batch, target_rgba);

	// Optimizer
	optimizer_step(stream);
}

void NerfNetwork::inference(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	float* pos_batch,
	float* dir_batch,
	// density network output must have space available for (color_network->input_width() * batch_size) elements of type network_precision_t
	network_precision_t* sigma,
	// color network output must have space available for (color_network->padded_output_width() * batch_size) elements of type network_precision_t
	network_precision_t* color
) {
	// Inference (density network)
	GPUMatrixDynamic density_network_input_matrix(
		pos_batch,
		density_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	GPUMatrixDynamic density_network_output_matrix(
		sigma,
		density_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	density_network->inference_mixed_precision(
		stream,
		density_network_input_matrix,
		density_network_output_matrix
	);

	// dir_batch or color can be nullptr if we only want to run inference on the density network
	if (dir_batch == nullptr || color == nullptr) {
		return;
	}

	// Inference (direction encoding)
	network_precision_t* direction_encoding_output = sigma + density_network->padded_output_width() * batch_size;

	GPUMatrixDynamic direction_encoding_input_matrix(
		dir_batch,
		direction_encoding->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	GPUMatrixDynamic direction_encoding_output_matrix(
		direction_encoding_output,
		direction_encoding->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	direction_encoding->inference_mixed_precision(
		stream,
		direction_encoding_input_matrix,
		direction_encoding_output_matrix
	);

	// Inference (color network)
	GPUMatrixDynamic color_network_input_matrix(
		density_network_output_matrix.data(),
		color_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	GPUMatrixDynamic color_network_output_matrix(
		color,
		color_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	color_network->inference_mixed_precision(
		stream,
		color_network_input_matrix,
		color_network_output_matrix
	);
}

std::unique_ptr<NerfNetwork::ForwardContext> NerfNetwork::forward(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	float* pos_batch,
	float* dir_batch,
	network_precision_t* network_sigma,
	network_precision_t* network_color
) {
	auto fwd_ctx = std::make_unique<ForwardContext>();

	// Forward pass on density network (with multiresolution hash encoding built in!)

	fwd_ctx->density_network_input_matrix = GPUMatrixDynamic(
		pos_batch,								// density network takes the sample positions as input
		density_network->input_width(),			// rows
		batch_size,								// cols
		MatrixLayout::RowMajor
	);

	// Here we make the output of the density network a pointer to the first half of the color network's input buffer.
	fwd_ctx->density_network_output_matrix = GPUMatrixDynamic(
		network_sigma,				 			// density network output = color network input
		density_network->output_width(), 		// rows
		batch_size,								// cols
		MatrixLayout::RowMajor
	);

	fwd_ctx->density_ctx = density_network->forward(
		stream,
		fwd_ctx->density_network_input_matrix,
		&fwd_ctx->density_network_output_matrix,
		false,
		true // prepare_input_gradients must be `true` otherwise backwards() fails (forward->dy_dx is not defined)
	);

	// Encode directions (dir_batch)
	// Direction encoding gets concatenated with density_network_output (which will just be the second half of network_sigma)

	network_precision_t* direction_encoding_output = network_sigma + density_network->padded_output_width() * batch_size;

	fwd_ctx->direction_encoding_input_matrix = GPUMatrixDynamic(
		dir_batch,									// pointer to source data
		direction_encoding->input_width(),			// rows
		batch_size,									// cols
		MatrixLayout::RowMajor
	);

	fwd_ctx->direction_encoding_output_matrix = GPUMatrixDynamic(
		direction_encoding_output,					// pointer to destination data
		direction_encoding->padded_output_width(),	// rows
		batch_size,									// cols
		MatrixLayout::RowMajor
	);

	direction_encoding->forward(
		stream,
		fwd_ctx->direction_encoding_input_matrix,
		&fwd_ctx->direction_encoding_output_matrix
	);

	// Perform the forward pass on the color network

	fwd_ctx->color_network_input_matrix = GPUMatrixDynamic(
		network_sigma,							// pointer to source data
		color_network->input_width(),			// matrix rows
		batch_size,								// matrix columns
		MatrixLayout::RowMajor
	);

	fwd_ctx->color_network_output_matrix = GPUMatrixDynamic(
		network_color,							// pointer to destination data
		color_network->padded_output_width(),	// matrix rows
		batch_size,								// matrix columns
		MatrixLayout::RowMajor
	);

	fwd_ctx->color_ctx = color_network->forward(
		stream,
		fwd_ctx->color_network_input_matrix,
		&fwd_ctx->color_network_output_matrix,
		false,
		true // prepare_input_gradients
	);

	return fwd_ctx;
}

float NerfNetwork::calculate_loss(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	const uint32_t* ray_steps,
	const uint32_t* ray_steps_cum,
	const float* sample_dt,
	const float* target_rgba,
	const network_precision_t* network_sigma,
	const network_precision_t* network_color
) {

	float n_raysf = n_rays;

	/**
	 * The density MLP maps the hash encoded position y = enc(x; 𝜃)
	 * to 16 output values, the first of which we treat as log-space density
	 * https://arxiv.org/abs/2201.05989 - Muller, et al. page 9
	 * 
	 */

	accumulate_ray_colors_from_samples_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_steps_cum,
		network_sigma,
		network_color,
		sample_dt,
		workspace.ray_rgba,
		workspace.trans_buf,
		workspace.alpha_buf,
		workspace.weight_buf
	);
	
	// Calculate sum of squared errors loss per ray
	calculate_sse_loss_per_ray_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_steps_cum,
		workspace.ray_rgba,
		target_rgba,
		workspace.loss_buf,
		workspace.pxdiff_buf
	);
	
	// Calculate gradients
	calculate_network_output_gradient<<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		batch_size,
		1.0f / (2.0f * n_raysf),
		network_color,
		workspace.pxdiff_buf,
		sample_dt,
		workspace.trans_buf,
		workspace.alpha_buf,
		workspace.weight_buf,
		LOSS_SCALE,
		workspace.grad_buf
	);

	// Add all loss values together
	thrust::device_ptr<float> loss_buffer_ptr(workspace.loss_buf);

	float sum_of_squared_pixel_errors = thrust::reduce(
		thrust::cuda::par_nosync.on(stream),
		loss_buffer_ptr,
		loss_buffer_ptr + n_rays,
		0.0f,
		thrust::plus<float>()
	);

	// Return mean loss
	return sum_of_squared_pixel_errors / (4.0f * n_raysf);
}

void NerfNetwork::backward(
	hipStream_t stream,
	std::unique_ptr<NerfNetwork::ForwardContext>& fwd_ctx,
	uint32_t batch_size,
	float* pos_batch,
	float* dir_batch,
	float* target_rgba
) {
	// Backpropagate through the color network
	GPUMatrixDynamic color_network_dL_doutput_matrix(
		workspace.grad_buf,
		color_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	GPUMatrixDynamic color_network_dL_dinput_matrix(
		workspace.color_network_dL_dinput,
		color_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	color_network->backward(
		stream,
		*fwd_ctx->color_ctx,
		fwd_ctx->color_network_input_matrix,
		fwd_ctx->color_network_output_matrix,
		color_network_dL_doutput_matrix,
		&color_network_dL_dinput_matrix
	);

	// Backpropagate through the density network
	GPUMatrixDynamic density_network_dL_dinput_matrix(
		workspace.density_network_dL_dinput,
		density_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	// Construct a dL_dinput matrix of the correct size
	// color_network_dL_dinput_matrix is too large since it is the concatenation of density's outputs and encoded directions

	GPUMatrixDynamic density_network_dL_doutput_matrix(
		color_network_dL_dinput_matrix.data(),
		density_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	// overwrite density_network_dL_doutput with manually calculated gradient

	CUDA_CHECK_THROW(
		hipMemcpyAsync(
			density_network_dL_doutput_matrix.data(),
			workspace.grad_buf + 3 * batch_size,
			sizeof(network_precision_t) * batch_size,
			hipMemcpyDeviceToDevice,
			stream
		)
	);

	density_network->backward(
		stream,
		*fwd_ctx->density_ctx,
		fwd_ctx->density_network_input_matrix,
		fwd_ctx->density_network_output_matrix,
		density_network_dL_doutput_matrix,
		&density_network_dL_dinput_matrix
	);

}

void NerfNetwork::optimizer_step(const hipStream_t& stream) {

	density_optimizer->step(
		stream,
		LOSS_SCALE,
		params_workspace.density_network_params_fp,
		params_workspace.density_network_params_hp,
		params_workspace.density_network_gradients_hp
	);

	color_optimizer->step(
		stream,
		LOSS_SCALE,
		params_workspace.color_network_params_fp,
		params_workspace.color_network_params_hp,
		params_workspace.color_network_gradients_hp
	);
}

// Only enlarge buffers needed for inference
void NerfNetwork::enlarge_workspace_if_needed(const hipStream_t& stream, const uint32_t& batch_size) {
	if (batch_size <= this->batch_size) {
		return;
	}

	workspace.enlarge(
		stream,
		batch_size,
		density_network->input_width(),
		density_network->padded_output_width(),
		direction_encoding->input_width(),
		direction_encoding->padded_output_width(),
		color_network->input_width(),
		color_network->padded_output_width()
	);

	this->batch_size = batch_size;
}
