#include "hip/hip_runtime.h"
// This code was adapted from nerfstudio (Copyright 2022 The Nerfstudio Team. All rights reserved.)
// https://github.com/nerfstudio-project/nerfstudio/blob/main/nerfstudio/fields/instant_ngp_field.py
// Please see LICENSES/nerfstudio-project_nerfstudio.md for license details.

#include <json/json.hpp>
#include <math.h>
#include <tiny-cuda-nn/common.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "../utils/gpu-image.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/training-network-kernels.cuh"
#include "nerf-network.h"

using namespace tcnn;
using namespace nrc;
using json = nlohmann::json;


#if TCNN_HALF_PRECISION
    constexpr float LOSS_SCALE = 128.0f;
#else
    constexpr float LOSS_SCALE = 1.0f;
#endif


// Constructor

NerfNetwork::NerfNetwork(const float& aabb_size) {
	this->aabb_size = aabb_size;

	// TODO: set this properly based on the aabb
	double per_level_scale = 1.4472692012786865;

	// Create the Direction Encoding
	json direction_encoding_config = {
		{"otype", "SphericalHarmonics"},
		{"degree", 4},
	};

	direction_encoding.reset(
		create_encoding<network_precision_t>(3, direction_encoding_config)
	);

	// Create the Density MLP
	
	json density_encoding_config = {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", per_level_scale},
	};

	json density_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 1},
	};

	density_network.reset(
		new NetworkWithInputEncoding<network_precision_t>(
			3,	// input dims
			16, // output dims
			density_encoding_config,
			density_network_config
		)
	);

	// Create the Color MLP

	uint32_t color_network_in_dim = direction_encoding->padded_output_width() + density_network->padded_output_width();

	const json color_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
		{"n_input_dims", color_network_in_dim},
		{"n_output_dims", 3},
	};

	color_network.reset(
		create_network<network_precision_t>(color_network_config)
	);
}

// initialize params and gradients for the networks (I have no idea if this is correct)
void NerfNetwork::prepare_for_training(const hipStream_t& stream) {

	size_t rng_seed = 72791;
	pcg32 rng(rng_seed);

	// concatenated network params and gradients
	uint32_t n_total_params = density_network->n_params() + color_network->n_params();

	params_fp.enlarge(n_total_params);
	params_hp.enlarge(n_total_params);

	gradients_hp.enlarge(n_total_params);
	gradients_hp.memset(0);

	// initialize params

	density_network->initialize_params(rng, params_fp.data());

	color_network->initialize_params(rng, params_fp.data() + density_network->n_params());

	// initialize_params only initializes full precision params, need to copy to half precision

	copy_and_cast<network_precision_t, float>(stream, n_total_params, params_hp.data(), params_fp.data());

	// assign params pointers

	density_network->set_params(
		params_hp.data(),
		params_hp.data(),
		gradients_hp.data()
	);

	color_network->set_params(
		params_hp.data() + density_network->n_params(),
		params_hp.data() + density_network->n_params(),
		gradients_hp.data() + density_network->n_params()
	);

	// initialize optimizers
	
	json optimizer_config = {
		{"otype", "Adam"},
		{"learning_rate", 1e-2},
		{"epsilon", 1e-15},
	};

	density_optimizer.reset(
		create_optimizer<network_precision_t>(optimizer_config)
	);

	density_optimizer->allocate(density_network);

	color_optimizer.reset(
		create_optimizer<network_precision_t>(optimizer_config)
	);
	
	color_optimizer->allocate(color_network);

	// flag for training enabled
	can_train = true;
}

void NerfNetwork::train(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	uint32_t* ray_steps,
	uint32_t* ray_steps_cum,
	float* pos_batch,
	float* dir_batch,
	float* dt_batch,
	float* target_rgba
) {
	enlarge_workspace_if_needed(stream, batch_size);

	// Normalize input for neural network
	generate_normalized_network_input(stream, batch_size, pos_batch, dir_batch, dt_batch);

	// Forward
	auto fwd_ctx = forward(stream, batch_size, workspace.normal_pos_batch, workspace.normal_dir_batch);

	// Loss
	float mse_loss = calculate_loss(
		stream,
		batch_size,
		n_rays,
		n_samples,
		ray_steps,
		ray_steps_cum,
		workspace.normal_dt_batch,
		target_rgba
	);

	printf("Loss: %f\n", mse_loss);

	// Backward
	backward(stream, fwd_ctx, batch_size, workspace.normal_pos_batch, workspace.normal_dir_batch, target_rgba);

	// Optimizer
	optimizer_step(stream);
}

// Normalizes input and saves it to the correct buffers (thank you @buriedanimal)
void NerfNetwork::generate_normalized_network_input(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const float* pos_batch,
	const float* dir_batch,
	const float* dt_batch
) {
	// Normalize input for neural network
	normalize_network_input_kernel<<<n_blocks_linear(batch_size), n_threads_linear, 0, stream>>>(
		batch_size,
		1.0f / aabb_size,
		pos_batch,
		dir_batch,
		dt_batch,
		workspace.normal_pos_batch,
		workspace.normal_dir_batch,
		workspace.normal_dt_batch
	);
}

void NerfNetwork::inference(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	float* pos_batch,
	float* dir_batch,
	// density network output must have space available for (color_network->input_width() * batch_size) elements of type network_precision_t
	network_precision_t* sigma,
	// color network output must have space available for (color_network->padded_output_width() * batch_size) elements of type network_precision_t
	network_precision_t* color
) {
	enlarge_workspace_if_needed(stream, batch_size);

	// Normalize input
	generate_normalized_network_input(stream, batch_size, pos_batch, dir_batch);

	// Inference (density network)
	GPUMatrix density_network_input_matrix(
		workspace.normal_pos_batch,
		density_network->input_width(),
		batch_size
	);

	GPUMatrix density_network_output_matrix(
		sigma,
		density_network->padded_output_width(),
		batch_size
	);

	density_network->inference_mixed_precision(
		stream,
		density_network_input_matrix,
		density_network_output_matrix
	);

	// Inference (direction encoding)
	network_precision_t* direction_encoding_output = color + density_network->output_width() * batch_size;

	GPUMatrix direction_encoding_input_matrix(
		workspace.normal_dir_batch,
		direction_encoding->input_width(),
		batch_size
	);

	GPUMatrix direction_encoding_output_matrix(
		direction_encoding_output,
		direction_encoding->padded_output_width(),
		batch_size
	);

	direction_encoding->inference_mixed_precision(
		stream,
		direction_encoding_input_matrix,
		direction_encoding_output_matrix
	);

	// Inference (color network)
	GPUMatrix color_network_input_matrix(
		density_network_output_matrix.data(),
		color_network->input_width(),
		batch_size
	);

	GPUMatrix color_network_output_matrix(
		color,
		color_network->padded_output_width(),
		batch_size
	);

	color_network->inference_mixed_precision(
		stream,
		color_network_input_matrix,
		color_network_output_matrix
	);
}

std::unique_ptr<NerfNetwork::ForwardContext> NerfNetwork::forward(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	float* pos_batch,
	float* dir_batch
) {
	auto fwd_ctx = std::make_unique<ForwardContext>();

	// Forward pass on density network (with multiresolution hash encoding built in!)

	fwd_ctx->density_network_input_matrix = GPUMatrix<float>(
		pos_batch,								// density network takes the sample positions as input
		density_network->input_width(),			// rows
		batch_size								// cols
	);

	// Here we make the output of the density network a pointer to the first half of the color network's input buffer.
	fwd_ctx->density_network_output_matrix = GPUMatrix<network_precision_t>(
		workspace.color_network_input, 			// density network output = color network input
		density_network->output_width(), 		// rows
		batch_size								// cols
	);

	fwd_ctx->density_ctx = density_network->forward(
		stream,
		fwd_ctx->density_network_input_matrix,
		&fwd_ctx->density_network_output_matrix,
		false,
		true // prepare_input_gradients must be `true` otherwise backwards() fails (forward->dy_dx is not defined)
	);

	// Encode directions (dir_batch)
	// Direction encoding gets concatenated with density_network_output (which will just be the second half of color_network_input)
	
	network_precision_t* direction_encoding_output = workspace.color_network_input + density_network->output_width() * batch_size;

	fwd_ctx->direction_encoding_input_matrix = GPUMatrix<float>(
		dir_batch,									// pointer to source data
		direction_encoding->input_width(),			// rows
		batch_size									// cols
	);

	fwd_ctx->direction_encoding_output_matrix = GPUMatrix<network_precision_t>(
		direction_encoding_output,					// pointer to destination data
		direction_encoding->padded_output_width(),	// rows
		batch_size									// cols
	);

	direction_encoding->inference_mixed_precision(
		stream,
		fwd_ctx->direction_encoding_input_matrix,
		fwd_ctx->direction_encoding_output_matrix
	);

	// Perform the forward pass on the color network

	fwd_ctx->color_network_input_matrix = GPUMatrix<network_precision_t>(
		workspace.color_network_input,				// pointer to source data
		color_network->input_width(),			// matrix rows
		batch_size								// matrix columns
	);

	fwd_ctx->color_network_output_matrix = GPUMatrix<network_precision_t>(
		workspace.color_network_output,			// pointer to destination data
		color_network->padded_output_width(),	// matrix rows
		batch_size								// matrix columns
	);

	fwd_ctx->color_ctx = color_network->forward(
		stream,
		fwd_ctx->color_network_input_matrix,
		&fwd_ctx->color_network_output_matrix,
		false,
		true // prepare_input_gradients
	);

	return fwd_ctx;
}

float NerfNetwork::calculate_loss(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	const uint32_t* ray_steps,
	const uint32_t* ray_steps_cum,
	const float* sample_dt,
	const float* target_rgba
) {

	float n_raysf = n_rays;

	/**
	 * The density MLP maps the hash encoded position y = enc(x; 𝜃)
	 * to 16 output values, the first of which we treat as log-space density
	 * https://arxiv.org/abs/2201.05989 - Muller, et al. page 9
	 * 
	 */
	const tcnn::network_precision_t* log_space_density = workspace.color_network_input;

	accumulate_ray_colors_from_samples_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_steps_cum,
		workspace.color_network_output,
		log_space_density,
		sample_dt,
		workspace.ray_rgba,
		workspace.trans_buf,
		workspace.alpha_buf,
		workspace.weight_buf
	);

	// Calculate mean-squared loss per ray
	calculate_sse_loss_per_ray_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_steps_cum,
		workspace.ray_rgba,
		target_rgba,
		workspace.loss_buf,
		workspace.pxdiff_buf
	);

	
	// Zero out the gradient buffer
	CUDA_CHECK_THROW(
		hipMemsetAsync(
			workspace.grad_buf,
			0,
			sizeof(network_precision_t) * color_network->padded_output_width() * batch_size,
			stream
		)
	);
	
	// Calculate gradients
	calculate_network_output_gradient<<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		batch_size,
		1.0f / (2.0f * n_raysf),
		workspace.color_network_output,
		workspace.pxdiff_buf,
		sample_dt,
		workspace.trans_buf,
		workspace.alpha_buf,
		workspace.weight_buf,
		LOSS_SCALE,
		workspace.grad_buf
	);

	// Add all loss values together
	thrust::device_ptr<float> loss_buffer_ptr(workspace.loss_buf);

	float sum_of_squared_pixel_errors = thrust::reduce(
		thrust::cuda::par_nosync.on(stream),
		loss_buffer_ptr,
		loss_buffer_ptr + n_rays,
		0.0f,
		thrust::plus<float>()
	);

	// Return mean loss
	return sum_of_squared_pixel_errors / (4.0f * n_raysf);
}

void NerfNetwork::backward(
	hipStream_t stream,
	std::unique_ptr<NerfNetwork::ForwardContext>& fwd_ctx,
	uint32_t batch_size,
	float* pos_batch,
	float* dir_batch,
	float* target_rgba
) {
	// Backpropagate through the color network
	GPUMatrix<network_precision_t> color_network_dL_doutput_matrix(
		workspace.grad_buf,
		color_network->padded_output_width(),
		batch_size
	);

	GPUMatrix<network_precision_t> color_network_dL_dinput_matrix(
		workspace.color_network_dL_dinput,
		color_network->input_width(),
		batch_size
	);

	color_network->backward(
		stream,
		*fwd_ctx->color_ctx,
		fwd_ctx->color_network_input_matrix,
		fwd_ctx->color_network_output_matrix,
		color_network_dL_doutput_matrix,
		&color_network_dL_dinput_matrix
	);

	// Backpropagate through the density network
	GPUMatrix<float> density_network_dL_dinput_matrix(
		workspace.density_network_dL_dinput,
		density_network->input_width(),
		batch_size
	);

	// Construct a dL_dinput matrix of the correct size
	// color_network_dL_dinput_matrix is too large since it is the concatenation of density's outputs and encoded directions

	GPUMatrix<network_precision_t> density_network_dL_doutput_matrix(
		color_network_dL_dinput_matrix.data(),
		density_network->padded_output_width(),
		batch_size
	);

	density_network->backward(
		stream,
		*fwd_ctx->density_ctx,
		fwd_ctx->density_network_input_matrix,
		fwd_ctx->density_network_output_matrix,
		density_network_dL_doutput_matrix,
		&density_network_dL_dinput_matrix
	);

}

void NerfNetwork::optimizer_step(const hipStream_t& stream) {

	density_optimizer->step(
		stream,
		LOSS_SCALE,
		params_fp.data(),
		params_hp.data(),
		gradients_hp.data()
	);

	color_optimizer->step(
		stream,
		LOSS_SCALE,
		params_fp.data() + density_network->n_params(),
		params_hp.data() + density_network->n_params(),
		gradients_hp.data() + density_network->n_params()
	);
}

// Only enlarge buffers needed for inference
void NerfNetwork::enlarge_workspace_if_needed(const hipStream_t& stream, const uint32_t& batch_size) {
	if (batch_size <= this->batch_size) {
		return;
	}

	workspace.enlarge(
		stream,
		batch_size,
		density_network->input_width(),
		density_network->padded_output_width(),
		direction_encoding->input_width(),
		direction_encoding->padded_output_width(),
		color_network->input_width(),
		color_network->padded_output_width(),
		can_train
	);

	this->batch_size = batch_size;
}