#include <hip/hip_runtime.h>

#include "../common.h"

#include "training-image.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include <stbi/stb_image.h>

using namespace turbo;

void TrainingImage::load_cpu() {
	data_cpu.reset(stbi_load(file_path.c_str(), &dimensions.x, &dimensions.y, &channels, 4));
}
