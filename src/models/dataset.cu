#include "hip/hip_runtime.h"
#include <atomic>
#include <cstddef>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <thread>

#include "dataset.h"
#include "../math/transform4f.cuh"

using namespace std;
using namespace filesystem;
using json = nlohmann::json;
using namespace turbo;

Dataset::Dataset(const string& file_path)
    : file_path(file_path)
{
    ifstream input_file(file_path);
    json json_data;
    input_file >> json_data;

	uint32_t n_frames = json_data["frames"].size();

    cameras.reserve(n_frames);
    images.reserve(n_frames);

    float w = json_data.value("w", 0.0f);
    float h = json_data.value("h", 0.0f);

    image_dimensions = make_int2((int)w, (int)h);
    n_pixels_per_image = (uint32_t)(w * h);

    float cx = json_data.value("cx", 0.5f * w);
    float cy = json_data.value("cy", 0.5f * h);
    float2 principal_point = make_float2(cx, cy);
    
    // TODO: per-camera focal length
    // if "fl_x" and "fl_y" are specified, these values are the focal lengths for their respective axes (in pixels)
    float fl_x, fl_y;
    if (json_data.contains("fl_x")) {
        fl_x = json_data["fl_x"];
        fl_y = json_data.value("fl_y", fl_x);
    } else if (json_data.contains("camera_angle_x")) {
        float ca_x = json_data["camera_angle_x"];
        float ca_y = json_data.value("camera_angle_y", ca_x);
        fl_x = 0.5f * w / tanf(0.5f * ca_x);
        fl_y = 0.5f * h / tanf(0.5f * ca_y);
    }
    
    float2 focal_length = make_float2(fl_x, fl_y);

    uint32_t aabb_size = std::min(json_data.value("aabb_scale", 16), 128);
    float scene_scale = json_data.value("scene_scale", 1.0f);

    bounding_box = BoundingBox((float)aabb_size);

    float global_near = json_data.value("near", 0.05f);
    float global_far = json_data.value("far", 128.0f);

    DistortionParams dist_params(
        json_data.value("k1", 0.0f),
        json_data.value("k2", 0.0f),
        json_data.value("k3", 0.0f),
        json_data.value("p1", 0.0f),
        json_data.value("p2", 0.0f)
    );

    path base_dir = path(file_path).parent_path(); // get the parent directory of file_path

    for (json frame : json_data["frames"]) {
        float near = scene_scale * frame.value("near", global_near);
        float far = scene_scale * frame.value("far", global_far);

        Transform4f transform_matrix(frame["transform_matrix"]);

        Transform4f camera_matrix = Transform4f::Scale(scene_scale) * transform_matrix.from_nerf();

        // TODO: per-camera dimensions
        cameras.emplace_back(
            image_dimensions,
            near,
            far,
            focal_length,
            principal_point,
            float2{0.0f, 0.0f},
            camera_matrix,
            dist_params
        );

        // images
        string file_path = frame["file_path"];
        path absolute_path = base_dir / file_path; // construct the absolute path using base_dir

        // only add the image if it exists
        if (exists(absolute_path)) {
            images.emplace_back(absolute_path.string(), image_dimensions);
        }
    }

    // remove excess allocated images
    images.shrink_to_fit();
}

Dataset::Dataset(
    const BoundingBox& bounding_box,
    const vector<Camera>& cameras,
    const vector<TrainingImage>& images
)   : bounding_box(bounding_box)
    , cameras(cameras)
    , images(images)
{
    image_dimensions = images[0].dimensions;
    n_pixels_per_image = image_dimensions.x * image_dimensions.y;
}

// this method was written (mostly) by ChatGPT!
void Dataset::load_images_in_parallel(std::function<void(const size_t, const TrainingImage&)> post_load_image) {
    const size_t num_threads = std::thread::hardware_concurrency(); // get the number of available hardware threads

    std::vector<std::thread> threads;
    std::atomic<std::size_t> index{ 0 }; // atomic variable to track the next image to be loaded
    for (size_t i = 0; i < num_threads; ++i) {
        // create a new thread to load images
        threads.emplace_back([&] {
            std::size_t local_index;
            while ((local_index = index.fetch_add(1)) < images.size()) {
                images[local_index].load_cpu();
                if (post_load_image) {
                    post_load_image(local_index, images[local_index]);
                }
            }
        });
    }

    // wait for all threads to complete
    for (auto& thread : threads) {
        thread.join();
    }
}

json Dataset::to_json() const {
    json json_data;

    json_data["w"] = image_dimensions.x;
    json_data["h"] = image_dimensions.y;

    json_data["cx"] = cameras[0].principal_point.x;
    json_data["cy"] = cameras[0].principal_point.y;

    json_data["fl_x"] = cameras[0].focal_length.x;
    json_data["fl_y"] = cameras[0].focal_length.y;

    json_data["k1"] = cameras[0].dist_params.k1;
    json_data["k2"] = cameras[0].dist_params.k2;
    json_data["k3"] = cameras[0].dist_params.k3;
    json_data["p1"] = cameras[0].dist_params.p1;
    json_data["p2"] = cameras[0].dist_params.p2;

    json_data["aabb_scale"] = bounding_box.size_x;
    json_data["scene_scale"] = 1.0f;

    json frames = json::array();

    for (size_t i = 0; i < cameras.size(); ++i) {
        json frame;

        frame["near"] = cameras[i].near;
        frame["far"] = cameras[i].far;

        frame["transform_matrix"] = cameras[i].transform.to_nerf().to_matrix().to_json();

        path image_path(images[i].file_path);
        path relative_path = image_path.lexically_relative(file_path.value().parent_path());
        
        // convert to posix
        string path_string = relative_path.string();
        std::replace(path_string.begin(), path_string.end(), '\\', '/');

        // save as posix
        frame["file_path"] = path_string;

        frames.push_back(frame);
    }

    json_data["frames"] = frames;

    return json_data;
}
