#include "hip/hip_runtime.h"
#include <atomic>
#include <cstddef>
#include <filesystem>
#include <fstream>
#include <functional>
#include <iostream>
#include <string>
#include <thread>
#include <vector>

#include "dataset.h"
#include "../utils/coordinate-transformations.cuh"

using namespace std;
using namespace filesystem;
using json = nlohmann::json;
using namespace nrc;

Dataset::Dataset(string file_path) {
    ifstream input_file(file_path);
    json json_data;
    input_file >> json_data;

	uint32_t n_frames = json_data["frames"].size();

    cameras.reserve(n_frames);
    images.reserve(n_frames);

    image_dimensions = make_int2(json_data["w"], json_data["h"]);
    n_pixels_per_image = (uint32_t)image_dimensions.x * (uint32_t)image_dimensions.y;
    n_channels_per_image = 4;
    
    // TODO: per-camera focal length
    float2 focal_length{json_data["fl_x"], json_data["fl_y"]};
    float2 view_angle{json_data["camera_angle_x"], json_data["camera_angle_y"]};
    float2 angle_tans{tanf(view_angle.x), tanf(view_angle.y)};
    // sensor size is the size of the sensor at distance 1 from the camera's origin
    float2 sensor_size{
        2.0f * focal_length.x * tanf(0.5f * view_angle.x),
        2.0f * focal_length.y * tanf(0.5f * view_angle.y)
    };

    uint32_t aabb_size = std::min(json_data.value("aabb_size", 16), 128);
    bounding_box = BoundingBox((float)aabb_size);

    path base_dir = path(file_path).parent_path(); // get the parent directory of file_path

    for (json frame : json_data["frames"]) {
        float near = frame.value("near", 2.0f);
        float far = frame.value("far", 16.0f);

        Matrix4f transform_matrix(frame["transform_matrix"]);

        Matrix4f camera_matrix = nerf_to_nrc(transform_matrix);
        
        // TODO: per-camera dimensions
        float2 sens_size{
            near / focal_length.x * sensor_size.x,
            near / focal_length.y * sensor_size.y
        };

        cameras.emplace_back(near, far, focal_length, image_dimensions, sens_size, camera_matrix);

        // images
        string file_path = frame["file_path"];
        path absolute_path = base_dir / file_path; // construct the absolute path using base_dir
        images.emplace_back(absolute_path.string(), image_dimensions);
    }

}

// this method was written (mostly) by ChatGPT!
void Dataset::load_images_in_parallel(std::function<void(const size_t, const TrainingImage&)> post_load_image) {
    const size_t num_threads = std::thread::hardware_concurrency(); // get the number of available hardware threads

    std::vector<std::thread> threads;
    std::atomic<std::size_t> index{ 0 }; // atomic variable to track the next image to be loaded
    for (size_t i = 0; i < num_threads; ++i) {
        // create a new thread to load images
        threads.emplace_back([&] {
            std::size_t local_index;
            while ((local_index = index.fetch_add(1)) < images.size()) {
                images[local_index].load_cpu(n_channels_per_image);
                if (post_load_image) {
                    post_load_image(local_index, images[local_index]);
                }
            }
        });
    }

    // wait for all threads to complete
    for (auto& thread : threads) {
        thread.join();
    }
}
