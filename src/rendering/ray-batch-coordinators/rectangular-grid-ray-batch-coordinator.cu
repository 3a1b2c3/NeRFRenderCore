#include "hip/hip_runtime.h"
#include "../../models/camera.cuh"
#include "../../models/ray.h"
#include "../../models/ray-batch.cuh"
#include "../../utils/device-math.cuh"
#include "../../common.h"
#include "rectangular-grid-ray-batch-coordinator.cuh"

using namespace tcnn;

TURBO_NAMESPACE_BEGIN

__global__ void generate_rectangular_grid_of_rays_kernel(
    const int n_rays,
    const int stride,
    const int2 grid_offset, // offset in camera-space pixels of the grid's origin
    const int2 grid_size, // size in camera-space pixels of the grid's extent
    const int2 grid_resolution, // resolution (number of samples) across the grid
    const Camera* __restrict__ camera,
    const BoundingBox* __restrict__ bbox,
    float* __restrict__ pos,
    float* __restrict__ dir,
    float* __restrict__ idir,
    float* __restrict__ t,
    float* __restrict__ t_max,
    int* __restrict__ index,
    bool* __restrict__ alive
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_rays) {
        return;
    }

    int y = divide(i, grid_resolution.x);  // (i / grid_resolution.x)
    int x = i - y * grid_resolution.x;    // (i % grid_resolution.x) 

    // calculate x and y in grid space
    int gx = (float)grid_size.x * (float)x / (float)grid_resolution.x;
    int gy = (float)grid_size.y * (float)y / (float)grid_resolution.y;

    // get pixel indices
    int ix = gx + grid_offset.x;
    int iy = gy + grid_offset.y;

    fill_ray_buffers(i, stride, camera, bbox, ix, iy, pos, dir, idir, t, t_max, index, alive);
}

void RectangularGridRayBatchCoordinator::generate_rays(
    const Camera* camera,
    const BoundingBox* bbox,
    RayBatch& ray_batch,
    const hipStream_t& stream
) {
    generate_rectangular_grid_of_rays_kernel<<<n_blocks_linear(ray_batch.size), n_threads_linear, 0, stream>>>(
        ray_batch.size,
        ray_batch.stride,
        grid_offset,
        grid_size,
        grid_resolution,
        camera,
        bbox,
        ray_batch.pos,
        ray_batch.dir,
        ray_batch.idir,
        ray_batch.t,
        ray_batch.t_max,
        ray_batch.index,
        ray_batch.alive
    );
}

__global__ void copy_packed_rgba_rectangular_grid_kernel(
    const int n_grid_pixels,
    const int stride,
    const int output_width,
    const int2 grid_offset,
    const int2 grid_size,
    const int2 grid_resolution,
    const float* __restrict__ rgba_in,
    float* __restrict__ rgba_out
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_grid_pixels) {
        return;
    }

    // calculate x and y in local grid space (output pixel)
    int oy = divide(i, grid_size.x);
    int ox = i - oy * grid_size.x;

    // get the corresponding buffer index (input pixel)
    const int ix = divide(ox * grid_resolution.x, grid_size.x);
    const int iy = divide(oy * grid_resolution.y, grid_size.y);

    int i_in = ix + iy * grid_resolution.x;

    // calculate index in output buffer
    ox += grid_offset.x;
    oy += grid_offset.y;

    int i_out = 4 * (ox + oy * output_width);

    // copy packed pixels to output
    #pragma unroll
    for (int j = 0; j < 4; ++j) {
        rgba_out[i_out] = rgba_in[i_in];
        i_out += 1;
        i_in += stride;
    }
}

void RectangularGridRayBatchCoordinator::copy_packed(
    const int& n_rays,
    const int2& output_size,
    const int& output_stride,
    float* rgba_in,
    float* rgba_out,
    const hipStream_t& stream
) {
    const int n_output_pixels = output_size.x * output_size.y;
    copy_packed_rgba_rectangular_grid_kernel<<<n_blocks_linear(n_output_pixels), n_threads_linear, 0, stream>>>(
        n_output_pixels,
        output_stride,
        output_size.x,
        grid_offset,
        grid_size,
        grid_resolution,
        rgba_in,
        rgba_out
    );
}

TURBO_NAMESPACE_END
