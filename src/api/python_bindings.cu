#include "hip/hip_runtime.h"
#include <json/json.hpp>
#include <pybind11/functional.h>
#include <pybind11/numpy.h>
#include <pybind11/operators.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/stl/filesystem.h>
#include <pybind11_json/pybind11_json.hpp>
#include <string>
#include <vector>

#include "../controllers/nerf-rendering-controller.h"
#include "../controllers/nerf-training-controller.h"
#include "../integrations/blender.cuh"
#include "../math/transform4f.cuh"
#include "../math/matrix4f.cuh"
#include "../models/bounding-box.cuh"
#include "../models/camera.cuh"
#include "../models/dataset.h"
#include "../models/nerf-proxy.cuh"
#include "../models/render-request.cuh"
#include "../models/updatable-property.cuh"
#include "../render-targets/cuda-render-buffer.cuh"
#include "../render-targets/opengl-render-surface.cuh"
#include "../services/device-manager.cuh"
#include "../services/nerf-manager.cuh"
#include "../services/runtime-manager.cuh"
#include "pybind_cpp_utils.cuh"
#include "pybind_cuda.cuh"

namespace py = pybind11;

using namespace turbo;

// a macro for easily forwarding get/set calls in pybind to UpdatableProperty
// usage: .def_property(UPDATABLE(SomeClass, PropType, attribute_name))
#define UPDATABLE(CLASS, TYPE, ATTR) \
    #ATTR, \
    [](CLASS& obj) -> TYPE { return obj.ATTR.get(); }, \
    [](CLASS& obj, TYPE value) { obj.ATTR.set(value); }

// a utility function for easily binding UpdatableProperty<T>
template <typename T>
void bind_updatable(py::module& m, const std::string& class_name) {
    py::class_<UpdatableProperty<T>>(m, class_name.c_str())
        .def(py::init<>())
        .def("get", &UpdatableProperty<T>::get)
        .def("set", &UpdatableProperty<T>::set)
        .def("set_dirty", &UpdatableProperty<T>::set_dirty)
        .def("is_dirty", &UpdatableProperty<T>::is_dirty)
    ;
}


PYBIND11_MODULE(PyTurboNeRF, m) {
    /**
     * Global attributes
     * 
     */

    m.doc() = "TurboNeRF Python Bindings";
    m.attr("__version__") = "0.0.19";
    m.attr("__build__") = 0;

    /**
     * Global functions
     */

    m.def(
        "teardown",
        []() {
            DeviceManager::teardown();
        }
    );
    
    /**
     * Utility classes
     */

    py::class_<Matrix4f>(m, "Matrix4f", py::buffer_protocol())
        .def(
            py::init(
                [](py::array_t<float> arr) {
                    const bool is_2d = arr.ndim() == 2;
                    const bool is_4x4 = arr.shape(0) == 4 && arr.shape(1) == 4;
                    
                    if (!is_2d || !is_4x4) {
                        throw std::runtime_error("Invalid shape for Matrix4f");
                    }

                    auto buf = arr.request();
                    auto ptr = (float*)buf.ptr;
                    return Matrix4f{
                        ptr[0],  ptr[1],  ptr[2],  ptr[3],
                        ptr[4],  ptr[5],  ptr[6],  ptr[7],
                        ptr[8],  ptr[9],  ptr[10], ptr[11],
                        ptr[12], ptr[13], ptr[14], ptr[15]
                    };
                }
            ),
            py::arg("matrix")
        )
        .def_buffer([](Matrix4f &m) -> py::buffer_info {
            return py::buffer_info(
                m.data(),
                sizeof(float),
                py::format_descriptor<float>::format(),
                2,
                {4, 4},
                {sizeof(float) * 4, sizeof(float)}
            );
        })
    ;

    py::class_<Transform4f>(m, "Transform4f", py::buffer_protocol())
        .def(
            py::init(
                [](py::array_t<float> arr) {
                    const bool is_2d = arr.ndim() == 2;
                    const bool is_3x4 = arr.shape(0) == 3 && arr.shape(1) == 4;
                    const bool is_4x4 = arr.shape(0) == 4 && arr.shape(1) == 4;
                    
                    if (!is_2d || (!is_3x4 && !is_4x4)) {
                        throw std::runtime_error("Invalid shape for Transform4f");
                    }

                    auto buf = arr.request();
                    auto ptr = (float*)buf.ptr;
                    return Transform4f{
                        ptr[0], ptr[1], ptr[2],  ptr[3],
                        ptr[4], ptr[5], ptr[6],  ptr[7],
                        ptr[8], ptr[9], ptr[10], ptr[11]
                    };
                }
            ),
            py::arg("matrix")
        )
        .def_buffer([](Transform4f &t) -> py::buffer_info {
            return py::buffer_info(
                t.data(),
                sizeof(float),
                py::format_descriptor<float>::format(),
                2,
                {3, 4},
                {sizeof(float) * 4, sizeof(float)}
            );
        })
        .def("from_nerf", &Transform4f::from_nerf)
        .def("to_nerf", &Transform4f::to_nerf)
        .def("to_matrix", &Transform4f::to_matrix)
    ;

    /**
     * Model classes
     */

    py::class_<DistortionParams>(m, "DistortionParams")
        .def(
            py::init<float, float, float, float, float>(),
            py::arg("k1") = 0.0f,
            py::arg("k2") = 0.0f,
            py::arg("k3") = 0.0f,
            py::arg("p1") = 0.0f,
            py::arg("p2") = 0.0f
        )
        .def_readwrite("k1", &DistortionParams::k1)
        .def_readwrite("k2", &DistortionParams::k2)
        .def_readwrite("k3", &DistortionParams::k3)
        .def_readwrite("p1", &DistortionParams::p1)
        .def_readwrite("p2", &DistortionParams::p2)
    ;

    py::class_<Camera>(m, "Camera")
        .def(
            py::init<
                int2,
                float,
                float,
                float2,
                float2,
                float2,
                Transform4f,
                DistortionParams
            >(),
            py::arg("resolution"),
            py::arg("near"),
            py::arg("far"),
            py::arg("focal_length"),
            py::arg("principal_point"),
            py::arg("shift"),
            py::arg("transform"),
            py::arg("dist_params") = DistortionParams()
        )
        .def_readwrite("resolution", &Camera::resolution)
        .def_readwrite("near", &Camera::near)
        .def_readwrite("far", &Camera::far)
        .def_readwrite("focal_length", &Camera::focal_length)
        .def_readwrite("principal_point", &Camera::principal_point)
        .def_readwrite("shift", &Camera::shift)
        .def_readwrite("transform", &Camera::transform)
        .def_readwrite("dist_params", &Camera::dist_params)
        .def_readwrite("show_image_planes", &Camera::show_image_planes)
        .def(py::self == py::self)
        .def(py::self != py::self)
    ;

    py::class_<Dataset>(m, "Dataset")
        .def(
            py::init<const string&>(),
            py::arg("file_path")
        )
        .def("load_transforms", &Dataset::load_transforms)
        .def("copy", &Dataset::copy)
        .def("to_json", &Dataset::to_json)
        .def(
            "set_camera_at",
            [](Dataset& ds, int index, Camera& cam) {
                ds.cameras[index] = cam;
            }
        )
        .def_readwrite("file_path", &Dataset::file_path)
        .def_readwrite("cameras", &Dataset::cameras)
        .def_readwrite("bounding_box", &Dataset::bounding_box)
        .def_readonly("image_dimensions", &Dataset::image_dimensions)
        .def_readonly("contains_multiple_image_dims", &Dataset::contains_multiple_image_dims)
    ;
    
    py::class_<BoundingBox>(m, "BoundingBox")
        .def(py::init<>())
        .def(
            py::init<float>(),
            py::arg("size")
        )
        .def("size", &BoundingBox::size)
        .def(py::self == py::self)
        .def(py::self != py::self)
        .def_readwrite("max_x", &BoundingBox::max_x)
        .def_readwrite("max_y", &BoundingBox::max_y)
        .def_readwrite("max_z", &BoundingBox::max_z)
        .def_readwrite("min_x", &BoundingBox::min_x)
        .def_readwrite("min_y", &BoundingBox::min_y)
        .def_readwrite("min_z", &BoundingBox::min_z)
    ;

    bind_updatable<BoundingBox>(m, "UpdatableBoundingBox");
    bind_updatable<Transform4f>(m, "UpdatableTransform4f");

    py::class_<NeRFProxy>(m, "NeRF")
        .def("attach_dataset", &NeRFProxy::attach_dataset)
        .def("detach_dataset", &NeRFProxy::detach_dataset)
        .def("is_dirty", &NeRFProxy::is_dirty)
        .def("can_train", &NeRFProxy::can_train)
        .def("is_image_data_loaded", &NeRFProxy::is_image_data_loaded)
        .def_property(UPDATABLE(NeRFProxy, BoundingBox, render_bbox))
        .def_property(UPDATABLE(NeRFProxy, BoundingBox, training_bbox))
        .def_property(UPDATABLE(NeRFProxy, Transform4f, transform))
        .def_readwrite("is_visible", &NeRFProxy::is_visible)
        .def_readwrite("is_dataset_dirty", &NeRFProxy::is_dataset_dirty)
        .def_readonly("dataset", &NeRFProxy::dataset)
        .def_readonly("can_render", &NeRFProxy::can_render)
        .def_readonly("training_step", &NeRFProxy::training_step)
        .def_readonly("id", &NeRFProxy::id)
    ;

    py::enum_<RenderPattern>(m, "RenderPattern")
        .value("LinearChunks", RenderPattern::LinearChunks)
        .value("HexagonalGrid", RenderPattern::HexagonalGrid)
        .value("RectangularGrid", RenderPattern::RectangularGrid)
    ;

    py::class_<RenderTarget>(m, "RenderTarget")
        .def(
            "save_image",
            [](RenderTarget& rt, const string& file_path) {
                rt.save_image(file_path);
            },
            py::arg("file_path")
        )
        .def(
            "set_size",
            [](RenderTarget& rt, const uint32_t& width, const uint32_t& height) {
                rt.set_size(width, height);
            },
            py::arg("width"),
            py::arg("height")
        )
        .def_readonly("width", &RenderTarget::width)
        .def_readonly("height", &RenderTarget::height)
        .def("free", [](RenderTarget& rt) { rt.free(); })
    ;

    py::class_<CPURenderBuffer, RenderTarget>(m, "CPURenderBuffer")
        .def(py::init<>())
        .def("get_rgba", [](CPURenderBuffer& rb) {

            rb.synchronize();
            float* rgba = rb.get_rgba();
            
            const int width = rb.width;
            const int height = rb.height;
            
            py::array::StridesContainer strides = {
                sizeof(float) * 4 * width,
                sizeof(float) * 4,
                sizeof(float)
            };
            
            return py::array_t<float>({ height, width, 4 }, strides, rgba);
        })
    ;

    py::class_<CUDARenderBuffer, RenderTarget>(m, "CUDARenderBuffer")
        .def(py::init<>())
    ;

    py::class_<OpenGLRenderSurface, RenderTarget>(m, "OpenGLRenderSurface")
        .def(py::init<>())
    ;

    py::enum_<RenderFlags>(m, "RenderFlags", py::arithmetic())
        .value("Preview", RenderFlags::Preview)
        .value("Final", RenderFlags::Final)
        .def(py::self | py::self)
        .def(py::self & py::self)
        .def(py::self |= py::self)
        .def(py::self &= py::self)
    ;

    py::class_<RenderProperties>(m, "RenderProperties")
        .def(py::init<>())
        .def_readwrite("show_near_planes", &RenderProperties::show_near_planes)
        .def_readwrite("show_far_planes", &RenderProperties::show_far_planes)
    ;

    py::class_<RenderModifiers>(m, "RenderModifiers")
        .def(py::init<>())
        .def_readwrite("properties", &RenderModifiers::properties)
    ;

    py::class_<RenderRequest, std::shared_ptr<RenderRequest>>(m, "RenderRequest")
        .def(
            py::init<
                const Camera&,
                std::vector<NeRFProxy*>&,
                RenderTarget*,
                const RenderModifiers&,
                const RenderFlags&,
                OnCompleteCallback,
                OnProgressCallback,
                OnCancelCallback
            >(),
            py::arg("camera"),
            py::arg("nerfs"),
            py::arg("output"),
            py::arg("modifiers") = RenderModifiers(),
            py::arg("flags") = RenderFlags::Final,
            py::arg("on_complete") = nullptr,
            py::arg("on_progress") = nullptr,
            py::arg("on_cancel") = nullptr
        )
    ;

    /**
     * Controller classes
     */

    py::class_<NeRFRenderingController, std::shared_ptr<NeRFRenderingController>>(m, "Renderer")
        .def(
           py::init<const RenderPattern&, const uint32_t&>(),
           py::arg("pattern") = RenderPattern::LinearChunks,
           py::arg("batch_size") = 0
        )
        .def(
            "submit",
            &NeRFRenderingController::submit,
            py::arg("request")
        )
        .def_readwrite("min_step_size", &NeRFRenderingController::min_step_size)
    ;

    // TrainingController helpers

    py::class_<NeRFTrainingController::TrainingMetrics>(m, "TrainingMetrics")
        .def_readonly("step", &NeRFTrainingController::TrainingMetrics::step)
        .def_readonly("loss", &NeRFTrainingController::TrainingMetrics::loss)
        .def_readonly("n_rays", &NeRFTrainingController::TrainingMetrics::n_rays)
        .def_readonly("n_samples", &NeRFTrainingController::TrainingMetrics::n_samples)
    ;

    py::class_<NeRFTrainingController::OccupancyGridMetrics>(m, "OccupancyGridMetrics")
        .def_readonly("n_occupied", &NeRFTrainingController::OccupancyGridMetrics::n_occupied)
        .def_readonly("n_total", &NeRFTrainingController::OccupancyGridMetrics::n_total)
    ;

    // TrainingController class

    py::class_<NeRFTrainingController, std::shared_ptr<NeRFTrainingController>>(m, "Trainer")
        .def(
            py::init<NeRFProxy*>(),
            py::arg("nerf")
        )
        .def(
            "setup_data",
            &NeRFTrainingController::setup_data,
            py::arg("batch_size"),
            "Call this once before starting training."
        )
        .def(
            "teardown",
            &NeRFTrainingController::teardown,
            "Call this before destroying the controller."
        )
        .def(
            "reset_training",
            &NeRFTrainingController::reset_training,
            "Call this to clear the training data."
        )
        .def(
            "load_images",
            [](NeRFTrainingController& tc, py::object on_image_loaded) {
                // coded with a generous amount of help from GPT-4
                if (py::isinstance<py::function>(on_image_loaded)) {
                    // Capture a reference to the on_image_loaded function
                    auto on_image_loaded_func = std::make_shared<py::function>(on_image_loaded.cast<py::function>());

                    // Release the GIL before calling the C++ function
                    py::gil_scoped_release release;

                    tc.load_images([on_image_loaded_func](int n_loaded, int n_total) {
                        // Re-acquire the GIL when calling the Python function
                        py::gil_scoped_acquire acquire;
                        (*on_image_loaded_func)(n_loaded, n_total);
                    });
                } else {
                    tc.load_images();
                }
            },
            py::arg("on_image_loaded") = py::none()
        )
        .def(
            "update_occupancy_grid",
            &NeRFTrainingController::update_occupancy_grid,
            py::arg("training_step")
        )
        .def("train_step", &NeRFTrainingController::train_step)
        .def_readwrite("alpha_selection_threshold", &NeRFTrainingController::alpha_selection_threshold)
        .def_readwrite("alpha_selection_probability", &NeRFTrainingController::alpha_selection_probability)
        .def_readwrite("min_step_size", &NeRFTrainingController::min_step_size)
        .def_readwrite("use_distortion_loss", &NeRFTrainingController::use_distortion_loss)
        .def_readwrite("nerf", &NeRFTrainingController::proxy)
    ;

    /**
     * Integration Modules
     */

    /**
     * Blender is the only integration for now.
     * 
     */

    // TODO: This is a large module.  Consider defining it in a separate file.

    py::enum_<BlenderBridge::Event>(m, "BlenderBridgeEvent")
        .value("OnDestroyNeRF", BlenderBridge::Event::OnDestroyNeRF)
        .value("OnUpdateOccupancyGrid", BlenderBridge::Event::OnUpdateOccupancyGrid)
        .value("OnPreviewStart", BlenderBridge::Event::OnPreviewStart)
        .value("OnPreviewProgress", BlenderBridge::Event::OnPreviewProgress)
        .value("OnPreviewComplete", BlenderBridge::Event::OnPreviewComplete)
        .value("OnPreviewCancel", BlenderBridge::Event::OnPreviewCancel)
        .value("OnRenderStart", BlenderBridge::Event::OnRenderStart)
        .value("OnRenderProgress", BlenderBridge::Event::OnRenderProgress)
        .value("OnRenderComplete", BlenderBridge::Event::OnRenderComplete)
        .value("OnRenderCancel", BlenderBridge::Event::OnRenderCancel)
        .value("OnRequestRedraw", BlenderBridge::Event::OnRequestRedraw)
        .value("OnTrainingImageLoaded", BlenderBridge::Event::OnTrainingImageLoaded)
        .value("OnTrainingImagesLoadComplete", BlenderBridge::Event::OnTrainingImagesLoadComplete)
        .value("OnTrainingImagesLoadStart", BlenderBridge::Event::OnTrainingImagesLoadStart)
        .value("OnTrainingImagesUnloaded", BlenderBridge::Event::OnTrainingImagesUnloaded)
        .value("OnTrainingReset", BlenderBridge::Event::OnTrainingReset)
        .value("OnTrainingStart", BlenderBridge::Event::OnTrainingStart)
        .value("OnTrainingStop", BlenderBridge::Event::OnTrainingStop)
        .value("OnTrainingStep", BlenderBridge::Event::OnTrainingStep)
    ;

    py::class_<BlenderBridge>(m, "BlenderBridge")
        .def(py::init<>())
        // properties
        .def_readonly("previewer", &BlenderBridge::previewer)
        .def_readonly("renderer", &BlenderBridge::renderer)
        // training
        .def("get_trainer_for_nerf", &BlenderBridge::trainer_for_proxy, py::arg("nerf"), py::return_value_policy::reference)
        .def("get_nerf", &BlenderBridge::get_nerf, py::arg("nerf_id"), py::return_value_policy::reference)
        .def("get_nerfs", &BlenderBridge::get_nerfs, py::return_value_policy::reference)
        .def("create_nerf", &BlenderBridge::create_nerf, py::arg("dataset"), py::return_value_policy::reference)
        .def("clone_nerf", &BlenderBridge::clone_nerf, py::arg("nerf"), py::return_value_policy::reference)
        .def("destroy_nerf", &BlenderBridge::destroy_nerf, py::arg("nerf"))
        .def("load_nerf", &BlenderBridge::load_nerf, py::arg("path"), py::return_value_policy::reference)
        .def("save_nerf", &BlenderBridge::save_nerf, py::arg("nerf"), py::arg("path"))
        .def("can_load_training_images", &BlenderBridge::can_load_training_images, py::arg("nerf"))
        .def("load_training_images", &BlenderBridge::load_training_images, py::arg("nerf"))
        .def("can_any_nerf_train", &BlenderBridge::can_any_nerf_train)
        .def("is_training", &BlenderBridge::is_training)
        .def("start_training", &BlenderBridge::start_training)
        .def("stop_training", &BlenderBridge::stop_training)
        .def("is_training_enabled", &BlenderBridge::is_training_enabled, py::arg("nerf"))
        .def("enable_training", &BlenderBridge::enable_training, py::arg("nerf"))
        .def("disable_training", &BlenderBridge::disable_training, py::arg("nerf"))
        .def("reset_training", &BlenderBridge::reset_training, py::arg("nerf"))
        .def("unload_training_images", &BlenderBridge::unload_training_images)
        // rendering (final)
        .def("is_rendering", &BlenderBridge::is_rendering)
        .def("get_render_progress", &BlenderBridge::get_render_progress)
        .def("cancel_render", &BlenderBridge::cancel_render)
        .def(
            "request_render",
            &BlenderBridge::request_render,
            py::arg("camera"),
            py::arg("proxies"),
            py::arg("modifiers") = RenderModifiers()
        )
        .def("get_render_rgba", [](BlenderBridge& bb) {
            float* rgba = bb.get_render_rgba();
            std::size_t n_pixels = bb.get_render_n_pixels();
            return py::memoryview::from_buffer(
                (void*)rgba, // data
                sizeof(float), // size of one element
                py::format_descriptor<float>::value, // format
                { 4 * n_pixels }, // shape
                { 0 }, // strides,
                true
            );
        })
        .def("get_render_n_pixels", &BlenderBridge::get_render_n_pixels)
        .def(
            "resize_render_surface",
            &BlenderBridge::resize_render_surface,
            py::arg("width"),
            py::arg("height")
        )
        // rendering (preview)
        .def("is_previewing", &BlenderBridge::is_previewing)
        .def("get_preview_progress", &BlenderBridge::get_preview_progress)
        .def("cancel_preview", &BlenderBridge::cancel_preview)
        .def(
            "request_preview",
            &BlenderBridge::request_preview,
            py::arg("camera"),
            py::arg("proxies"),
            py::arg("flags"),
            py::arg("modifiers") = RenderModifiers()
        )
        .def(
            "resize_preview_surface",
            &BlenderBridge::resize_preview_surface,
            py::arg("width"),
            py::arg("height")
        )
        // drawing
        .def("enqueue_redraw", &BlenderBridge::enqueue_redraw)
        .def("draw", &BlenderBridge::draw)
        // event observers
        .def(
            "add_observer",
            [](BlenderBridge& bb, BlenderBridge::Event event, py::function callback) -> uint32_t {
                return bb.event_bus.add_observer(event, [callback](BlenderBridge::EventCallbackParam e) {
                    py::gil_scoped_acquire acquire;
                    callback(cpp_map_to_py_dict(e));
                });
            },
            py::arg("event"),
            py::arg("callback")
        )
        .def(
            "remove_observer",
            [](BlenderBridge& bb, uint32_t id) {
                bb.event_bus.remove_observer(id);
            },
            py::arg("id")
        )
    ;

    /**
     * Service classes
     */

    py::class_<NeRFManager>(m, "NeRFManager")
        .def(py::init<>())
        .def(
            "create",
            &NeRFManager::create,
            py::return_value_policy::reference
        )
        .def(
            "clone",
            &NeRFManager::clone,
            py::arg("nerf"),
            py::return_value_policy::reference
        )
        .def(
            "destroy",
            &NeRFManager::destroy,
            py::arg("nerf")
        )
    ;

    py::class_<FileManager>(m, "FileManager")
        .def_static(
            "load",
            [](NeRFProxy* nerf, std::string& path) {
                FileManager::load(nerf, path);
            },
            py::arg("nerf"),
            py::arg("path")
        )
        .def_static(
            "save",
            [](NeRFProxy* nerf, std::string& path) {
                FileManager::save(nerf, path);
            },
            py::arg("nerf"),
            py::arg("path")
        )
    ;

    /**
     * Runtime Utils (for checking CUDA compatibility)
     */

    py::class_<RuntimeVersion>(m, "RuntimeVersion")
        .def_readonly("major", &RuntimeVersion::major)
        .def_readonly("minor", &RuntimeVersion::minor)
        .def_readonly("subminor", &RuntimeVersion::subminor)
        .def("to_string", &RuntimeVersion::to_string)
        .def_static("CompiledRuntimeVersion", &RuntimeVersion::CompiledRuntimeVersion)
    ;

    py::class_<DeviceArchitecture>(m, "DeviceArchitecture")
        .def_readonly("major", &DeviceArchitecture::major)
        .def_readonly("minor", &DeviceArchitecture::minor)
        .def("to_string", &DeviceArchitecture::to_string)
        .def("arch_name", &DeviceArchitecture::arch_name)
    ;

    py::class_<RuntimeManager>(m, "RuntimeManager")
        .def(py::init<>())
        .def_readonly("cuda_archs", &RuntimeManager::cuda_archs)
        .def_static("required_runtime_version", &RuntimeManager::required_runtime_version)
        .def_static("current_runtime_version", &RuntimeManager::current_runtime_version)
        .def_static("is_driver_version_sufficient", &RuntimeManager::is_driver_version_sufficient)
        .def_static("get_device_architecture", &RuntimeManager::get_device_architecture)
        .def("check_runtime", &RuntimeManager::check_runtime)
    ;
}
