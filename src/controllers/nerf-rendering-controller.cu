#include "hip/hip_runtime.h"
#include <tiny-cuda-nn/common.h>
#include "nerf-rendering-controller.h"
#include "../models/camera.cuh"
#include "../utils/cu-compactor.cuh"
#include "../utils/nerf-constants.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/rendering-kernels.cuh"

using namespace nrc;
using namespace tcnn;

NeRFRenderingController::NeRFRenderingController(
    uint32_t batch_size
) {
    if (batch_size == 0) {
        // TODO: determine batch size from GPU specs
        this->batch_size = 1<<21;
    } else {
        this->batch_size = batch_size;
    }
}

void NeRFRenderingController::request_render(
    const hipStream_t& stream,
    const RenderRequest& request
) {
    // TODO: this should happen for all NeRFs
    NeRF* nerf = request.nerfs[0];

    // TODO: enlarge workspace only on batch size or output size change
    workspace.enlarge(
        stream,
        request.output.width,
        request.output.height,
        batch_size,
        nerf->network.get_concat_buffer_width(),
        nerf->network.get_padded_output_width(),
        n_threads_linear
    );

    printf("Rendering...\n");

    // workspace.camera = request.camera
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.camera,
            &request.camera,
            sizeof(Camera),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // workspace.bounding_box = nerf->bounding_box
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.bounding_box,
            &nerf->bounding_box,
            sizeof(BoundingBox),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // workspace.occupancy_grid = nerf->occupancy_grid
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.occupancy_grid,
            &nerf->occupancy_grid,
            sizeof(CascadedOccupancyGrid),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // calculate the number of pixels we need to fill
    uint32_t n_pixels = request.output.width * request.output.height;
    uint32_t n_rays_total = n_pixels;

    // double buffer indices
    int active_buf_idx = 0;
    int compact_buf_idx = 1;

    // loop over all pixels, chunked by batch size
    uint32_t n_pixels_filled = 0;
    while (n_pixels_filled < n_pixels) {
        // TODO:
        // for (auto& n in nerfs) { ... 
        // calculate the number of pixels to fill in this batch
        uint32_t n_pixels_to_fill = std::min(
            batch_size,
            n_pixels - n_pixels_filled
        );

        uint32_t n_rays = n_pixels_to_fill;

        // calculate the pixel indices to fill in this batch
        uint32_t pixel_start = n_pixels_filled;

        // generate rays for the pixels in this batch
        generate_rays_pinhole_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
            n_rays,
            batch_size,
            workspace.bounding_box,
            workspace.camera,
            workspace.ray_origin[active_buf_idx],
            workspace.ray_dir[active_buf_idx],
            workspace.ray_idir[active_buf_idx],
            workspace.ray_t[active_buf_idx],
            workspace.ray_idx[active_buf_idx],
            workspace.ray_alive,
            pixel_start
        );

        // initialize other ray properties

        // ray_sigma = 0
        CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_sigma[active_buf_idx], 0, batch_size * sizeof(float), stream));

        // ray_active = true
        CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_active[active_buf_idx], true, batch_size * sizeof(bool), stream));

        
        float dt_min = NeRFConstants::min_step_size;
        float dt_max = nerf->bounding_box.size_x * dt_min;
        const float cone_angle = NeRFConstants::cone_angle;

        // ray marching loop
        uint32_t n_rays_alive = n_rays;

        // TODO: march rays to bounding box first
        while (n_rays_alive > 0) {
            uint32_t network_batch = tcnn::next_multiple(n_rays_alive, tcnn::batch_size_granularity);

            // march each ray one step
            // TODO: should we march potentially multiple steps to maximize occupancy?
            march_rays_and_generate_network_inputs_kernel<<<n_blocks_linear(n_rays_alive), n_threads_linear, 0, stream>>>(
                n_rays_alive,
                batch_size,
                network_batch,
                workspace.occupancy_grid,
                workspace.bounding_box,
                1.0f / nerf->bounding_box.size_x,
                dt_min,
                dt_max,
                cone_angle,
                workspace.ray_origin[active_buf_idx],
                workspace.ray_dir[active_buf_idx],
                workspace.ray_idir[active_buf_idx],
                workspace.ray_alive,
                workspace.ray_active[active_buf_idx],
                workspace.ray_t[active_buf_idx],
                workspace.network_pos,
                workspace.network_dir,
                workspace.network_dt
            );

            // query the NeRF network for the samples
            nerf->network.inference(
                stream,
                network_batch,
                workspace.network_pos,
                workspace.network_dir,
                workspace.network_concat,
                workspace.network_output
            );

            // CHECK_DATA(color_cpu, float, workspace.network_color, network_batch * 3);

            // check how many color values in color_cpu are >1
            // uint32_t color_values_great_than_1 = 0;

            // for (uint32_t i = 0; i < network_batch * 3; i++) {
            //     if (color_cpu[i] > 1.0f) {
            //         color_values_great_than_1++;
            //     }
            // }

            // printf("# of color values >1: %d\n", color_values_great_than_1);

            // accumulate these samples into the pixel colors
            composite_samples_kernel<<<n_blocks_linear(n_rays_alive), n_threads_linear, 0, stream>>>(
                n_rays_alive,
                network_batch,
                request.output.stride,
                workspace.network_output,
                workspace.network_dt,
                workspace.ray_idx[active_buf_idx],
                workspace.ray_active[active_buf_idx],
                workspace.ray_alive,
                workspace.ray_sigma[active_buf_idx],
                request.output.rgba
            );

            // update how many rays are still alive
            const int n_rays_to_compact = calculate_block_counts_and_offsets(
                stream,
                n_rays_alive,
                workspace.c_block_size,
                workspace.ray_alive,
                workspace.c_block_counts,
                workspace.c_block_offsets
            );

            // if no rays are alive, we can skip compositing
            if (n_rays_to_compact == 0) {
                break;
            }
            
            // check if compaction is required
            if (n_rays_to_compact < n_rays_alive) {
                // get compacted ray indices
                generate_compaction_indices(
                    stream,
                    n_rays_alive,
                    workspace.c_block_size,
                    workspace.ray_alive,
                    workspace.c_block_offsets,
                    workspace.compact_idx
                );

                // compact ray properties via the indices
                compact_rays_kernel<<<n_blocks_linear(n_rays_to_compact), n_threads_linear, 0, stream>>>(
                    n_rays_to_compact,
                    batch_size,
                    workspace.compact_idx,

                    // input
                    workspace.ray_idx[active_buf_idx],
                    workspace.ray_active[active_buf_idx],
                    workspace.ray_t[active_buf_idx],
                    workspace.ray_origin[active_buf_idx],
                    workspace.ray_dir[active_buf_idx],
                    workspace.ray_idir[active_buf_idx],
                    workspace.ray_sigma[active_buf_idx],

                    // output
                    workspace.ray_idx[compact_buf_idx],
                    workspace.ray_active[compact_buf_idx],
                    workspace.ray_t[compact_buf_idx],
                    workspace.ray_origin[compact_buf_idx],
                    workspace.ray_dir[compact_buf_idx],
                    workspace.ray_idir[compact_buf_idx],
                    workspace.ray_sigma[compact_buf_idx]
                );

                

                // all compacted rays are alive
                CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_alive, true, n_rays_to_compact * sizeof(bool), stream));

                // swap the active and compact buffer indices
                std::swap(active_buf_idx, compact_buf_idx);

                // update n_rays_alive
                n_rays_alive = n_rays_to_compact;
            }
        }
        // increment the number of pixels filled
        n_pixels_filled += n_pixels_to_fill;
    }
};
