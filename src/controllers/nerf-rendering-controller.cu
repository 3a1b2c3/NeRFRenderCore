#include "hip/hip_runtime.h"
#include <tiny-cuda-nn/common.h>
#include "nerf-rendering-controller.h"
#include "../models/camera.cuh"
#include "../utils/nerf-constants.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/rendering-kernels.cuh"
#include "../utils/stream-compaction.cuh"

using namespace nrc;
using namespace tcnn;

NeRFRenderingController::NeRFRenderingController(
    uint32_t batch_size
) {
    if (batch_size == 0) {
        // TODO: determine batch size from GPU specs
        this->batch_size = 1<<21;
    } else {
        this->batch_size = batch_size;
    }
}

void NeRFRenderingController::request_render(
    const hipStream_t& stream,
    const RenderRequest& request
) {
    // TODO: this should happen for all NeRFs
    NeRF* nerf = request.nerfs[0];

    // TODO: enlarge workspace only on batch size or output size change
    workspace.enlarge(
        stream,
        request.output.width,
        request.output.height,
        batch_size,
        nerf->network.get_concat_buffer_width(),
        nerf->network.get_padded_output_width(),
        n_threads_linear
    );

    printf("Rendering...\n");

    // workspace.camera = request.camera
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.camera,
            &request.camera,
            sizeof(Camera),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // workspace.bounding_box = nerf->bounding_box
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.bounding_box,
            &nerf->bounding_box,
            sizeof(BoundingBox),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // workspace.occupancy_grid = nerf->occupancy_grid
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.occupancy_grid,
            &nerf->occupancy_grid,
            sizeof(CascadedOccupancyGrid),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // calculate the number of pixels we need to fill
    uint32_t n_pixels = request.output.width * request.output.height;

    // double buffer indices
    int active_buf_idx = 0;
    int compact_buf_idx = 1;

    // loop over all pixels, chunked by batch size
    uint32_t n_pixels_filled = 0;
    while (n_pixels_filled < n_pixels) {
        // TODO:
        // for (auto& n in nerfs) { ... 
        // calculate the number of pixels to fill in this batch
        uint32_t n_pixels_to_fill = std::min(
            batch_size,
            n_pixels - n_pixels_filled
        );

        uint32_t n_rays = n_pixels_to_fill;

        // calculate the pixel indices to fill in this batch
        uint32_t pixel_start = n_pixels_filled;

        // generate rays for the pixels in this batch
        generate_rays_pinhole_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
            n_rays,
            batch_size,
            workspace.bounding_box,
            workspace.camera,
            workspace.ray_origin[active_buf_idx],
            workspace.ray_dir[active_buf_idx],
            workspace.ray_idir[active_buf_idx],
            workspace.ray_t[active_buf_idx],
            workspace.ray_trans[active_buf_idx],
            workspace.ray_idx[active_buf_idx],
            workspace.ray_alive,
            workspace.ray_active[active_buf_idx],
            pixel_start
        );

        
        const float dt_min = NeRFConstants::min_step_size;
        const float dt_max = nerf->bounding_box.size_x * dt_min;
        const float cone_angle = NeRFConstants::cone_angle;

        // ray marching loop
        uint32_t n_rays_alive = n_rays;
        
        int n_steps = 0;

        // TODO: march rays to bounding box first
        while (n_rays_alive > 0) {

            // need to figure out how many rays can fit in this batch
            const uint32_t n_steps_per_ray = std::max(batch_size / n_rays_alive, (uint32_t)1);
            const uint32_t network_batch = tcnn::next_multiple(n_steps_per_ray * n_rays_alive, tcnn::batch_size_granularity);

            // march each ray one step
            march_rays_and_generate_network_inputs_kernel<<<n_blocks_linear(n_rays_alive), n_threads_linear, 0, stream>>>(
                n_rays_alive,
                batch_size,
                n_steps_per_ray,
                network_batch,
                workspace.occupancy_grid,
                workspace.bounding_box,
                1.0f / nerf->bounding_box.size_x,
                dt_min,
                dt_max,
                cone_angle,

                // input buffers
                workspace.ray_origin[active_buf_idx],
                workspace.ray_dir[active_buf_idx],
                workspace.ray_idir[active_buf_idx],
                workspace.ray_alive,
                workspace.ray_active[active_buf_idx],
                workspace.ray_t[active_buf_idx],

                // output buffers
                workspace.ray_steps[active_buf_idx],
                workspace.network_pos,
                workspace.network_dir,
                workspace.network_dt
            );

            // query the NeRF network for the samples
            nerf->network.inference(
                stream,
                network_batch,
                workspace.network_pos,
                workspace.network_dir,
                workspace.network_concat,
                workspace.network_output
            );

            // accumulate these samples into the pixel colors
            composite_samples_kernel<<<n_blocks_linear(n_rays_alive), n_threads_linear, 0, stream>>>(
                n_rays_alive,
                network_batch,
                request.output.stride,

                // input buffers
                workspace.ray_active[active_buf_idx],
                workspace.ray_steps[active_buf_idx],
                workspace.ray_idx[active_buf_idx],
                workspace.network_dt,
                workspace.network_output,
                
                // output buffers
                workspace.ray_alive,
                workspace.ray_trans[active_buf_idx],
                request.output.rgba
            );

            n_steps += n_steps_per_ray;
            if (n_steps < NeRFConstants::n_steps_per_render_compaction) {
                continue;
            }

            // update how many rays are still alive
            const int n_rays_to_compact = count_true_elements(
                stream,
                n_rays_alive,
                workspace.ray_alive
            );

            // if no rays are alive, we can skip compositing
            if (n_rays_to_compact == 0) {
                break;
            }
            
            // check if compaction is required
            if (n_rays_to_compact < n_rays_alive) {
                // get compacted ray indices
                generate_compaction_indices(
                    stream,
                    n_rays_alive,
                    workspace.ray_alive,
                    workspace.compact_idx
                );

                // compact ray properties via the indices
                compact_rays_kernel<<<n_blocks_linear(n_rays_to_compact), n_threads_linear, 0, stream>>>(
                    n_rays_to_compact,
                    batch_size,
                    workspace.compact_idx,

                    // input
                    workspace.ray_idx[active_buf_idx],
                    workspace.ray_active[active_buf_idx],
                    workspace.ray_t[active_buf_idx],
                    workspace.ray_origin[active_buf_idx],
                    workspace.ray_dir[active_buf_idx],
                    workspace.ray_idir[active_buf_idx],
                    workspace.ray_trans[active_buf_idx],

                    // output
                    workspace.ray_idx[compact_buf_idx],
                    workspace.ray_active[compact_buf_idx],
                    workspace.ray_t[compact_buf_idx],
                    workspace.ray_origin[compact_buf_idx],
                    workspace.ray_dir[compact_buf_idx],
                    workspace.ray_idir[compact_buf_idx],
                    workspace.ray_trans[compact_buf_idx]
                );

                // all compacted rays are alive
                CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_alive, true, n_rays_to_compact * sizeof(bool), stream));

                // swap the active and compact buffer indices
                std::swap(active_buf_idx, compact_buf_idx);

                printf("compacted %d rays to %d rays\n", n_rays_alive, n_rays_to_compact);

                // update n_rays_alive
                n_rays_alive = n_rays_to_compact;

                n_steps = 0;
            }
        }
        // increment the number of pixels filled
        n_pixels_filled += n_pixels_to_fill;
    }
};
