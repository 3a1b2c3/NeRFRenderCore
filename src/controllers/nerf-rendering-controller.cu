#include "hip/hip_runtime.h"
#include <tiny-cuda-nn/common.h>
#include "nerf-rendering-controller.h"
#include "../models/camera.cuh"
#include "../utils/cu-compactor.cuh"
#include "../utils/rendering-kernels.cuh"

using namespace nrc;
using namespace tcnn;

NeRFRenderingController::NeRFRenderingController(
    uint32_t batch_size
) {
    if (batch_size == 0) {
        // TODO: determine batch size from GPU specs
        this->batch_size = 1024;
    } else {
        this->batch_size = batch_size;
    }
}

void NeRFRenderingController::request_render(
    const hipStream_t& stream,
    const RenderRequest& request
) {
    // TODO: this should happen for all NeRFs
    NeRF* nerf = request.nerfs[0];

    // TODO: enlarge workspace only on batch size or output size change
    workspace.enlarge(
        stream,
        request.output.width,
        request.output.height,
        batch_size,
        nerf->network.get_color_network_input_width(),
        nerf->network.get_color_network_output_width()
    );

    printf("Rendering...\n");

    // workspace.camera = request.camera
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.camera,
            &request.camera,
            sizeof(Camera),
            hipMemcpyHostToDevice,
            stream
        )
    );


    // workspace.bounding_box = nerf->bounding_box
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.bounding_box,
            &nerf->bounding_box,
            sizeof(BoundingBox),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // workspace.occupancy_grid = nerf->occupancy_grid
    CUDA_CHECK_THROW(
        hipMemcpyAsync(
            workspace.occupancy_grid,
            &nerf->occupancy_grid,
            sizeof(CascadedOccupancyGrid),
            hipMemcpyHostToDevice,
            stream
        )
    );

    // calculate the number of pixels we need to fill
    uint32_t n_pixels = request.output.width * request.output.height;
    uint32_t n_rays_total = n_pixels;

    // loop over all pixels, chunked by batch size
    uint32_t n_pixels_filled = 0;
    while (n_pixels_filled < n_pixels) {
        // TODO:
        // for (auto& n in nerfs) { ... 
        // calculate the number of pixels to fill in this batch
        uint32_t n_pixels_to_fill = std::min(
            batch_size,
            n_pixels - n_pixels_filled
        );

        uint32_t n_rays = n_pixels_to_fill;

        // calculate the pixel indices to fill in this batch
        uint32_t pixel_start = n_pixels_filled;
        uint32_t pixel_end = pixel_start + n_pixels_to_fill;

        // generate rays for the pixels in this batch
        generate_rays_pinhole_kernel<<<n_blocks_linear(n_pixels_to_fill), n_threads_linear, 0, stream>>>(
            n_rays,
            batch_size,
            workspace.camera,
            workspace.ray_origin,
            workspace.ray_dir,
            workspace.ray_idir,
            workspace.ray_idx,
            pixel_start,
            pixel_end
        );

        CHECK_DATA(ray_idx_cpu, uint32_t, workspace.ray_idx, batch_size);

        CHECK_DATA(ray_origin_cpu, float, workspace.ray_origin, batch_size * 3);
        CHECK_DATA(ray_dir_cpu, float, workspace.ray_dir, batch_size * 3);
        CHECK_DATA(ray_idir_cpu, float, workspace.ray_idir, batch_size * 3);


        // initialize other ray properties
        // ray_t = 0
        CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_t, 0, batch_size * sizeof(float), stream));

        // ray_alive = true
        CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_alive, true, batch_size * sizeof(bool), stream));

        // ray_active = true
        CUDA_CHECK_THROW(hipMemsetAsync(workspace.ray_active, true, batch_size * sizeof(bool), stream));

        // TODO: figure out correct values for these
        const float dt_min = 0.01f;
        const float dt_max = 1.0f;
        const float cone_angle = 1.0f;

        // ray marching loop
        uint32_t n_rays_alive = n_rays;

        // TODO: march rays to bounding box first
        while (n_rays_alive > 0) {
            // march each ray one step
            // TODO: should we march potentially multiple steps to maximize occupancy?
            march_rays_and_generate_samples_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
                n_rays,
                batch_size,
                workspace.bounding_box,
                workspace.occupancy_grid,
                dt_min,
                dt_max,
                cone_angle,
                workspace.ray_origin,
                workspace.ray_dir,
                workspace.ray_idir,
                workspace.ray_alive,
                workspace.ray_active,
                workspace.ray_t,
                workspace.sample_pos,
                workspace.sample_dt
            );

            // query the NeRF network for the samples
            nerf->network.inference(
                stream,
                batch_size,
                workspace.sample_pos,
                workspace.ray_dir,
                workspace.network_sigma,
                workspace.network_color
            );

            // accumulate these samples into the pixel colors
            composite_samples_kernel<<<n_blocks_linear(n_pixels_to_fill), n_threads_linear, 0, stream>>>(
                n_rays,
                batch_size,
                workspace.network_sigma,
                workspace.network_color,
                workspace.sample_dt,
                workspace.ray_idx,
                workspace.ray_alive,
                workspace.ray_active,
                request.output.rgba
            );

            n_rays_alive = generate_compaction_indices(
                stream,
                batch_size,
                n_threads_linear,
                workspace.ray_alive,
                workspace.compact_idx
            );

            CHECK_DATA(compact_idx, int, workspace.compact_idx, batch_size);

            printf("n_rays_alive: %d\n", n_rays_alive);

        }
        // increment the number of pixels filled
        n_pixels_filled += n_pixels_to_fill;
    }
};
