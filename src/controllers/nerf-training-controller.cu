#include "hip/hip_runtime.h"

#include <>
#include <json/json.hpp>
#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/encoding.h>

#include <memory>
#include "nerf-training-controller.h"

using namespace nrc;
using namespace tcnn;
using namespace nlohmann;

NeRFTrainingController::NeRFTrainingController(
	const Dataset& dataset,
	const uint32_t& num_layers,
	const uint32_t& hidden_dim,
	const uint32_t& geo_feat_dim,
	const uint32_t& num_layers_color,
	const uint32_t& hidden_dim_color
) {
	this->dataset = dataset;
	this->num_layers = num_layers;
	this->hidden_dim = hidden_dim;
	this->geo_feat_dim = geo_feat_dim;
	this->num_layers_color = num_layers_color;
	this->hidden_dim_color = hidden_dim_color;
	
	// TODO: set this properly based on the aabb
	double per_level_scale = 1.4472692012786865;

	// Create the Direction Encoding
	json direction_encoding_config = {
		{"otype", "SphericalHarmonics"},
		{"degree", 4},
	};
	
	direction_encoding = std::shared_ptr<Encoding<network_precision_t>>(
		tcnn::create_encoding<network_precision_t>((uint32_t)3, direction_encoding_config)
	);
	
	// Create the Density MLP
	json density_mlp_encoding_config = {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", per_level_scale},
	};
	
	json density_mlp_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", hidden_dim},
		{"n_hidden_layers", num_layers - 1},
	};

	density_mlp = std::shared_ptr<tcnn::cpp::Module>(
		tcnn::cpp::create_network_with_input_encoding(3, 1 + geo_feat_dim, density_mlp_encoding_config, density_mlp_network_config)
	);
	
	// Create the Color MLP
	
	uint32_t color_mlp_in_dim = direction_encoding->padded_output_width() + geo_feat_dim;
		
	json color_mlp_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", hidden_dim_color},
		{"n_hidden_layers", num_layers_color - 1},
	};
	
	color_mlp = std::shared_ptr<tcnn::cpp::Module>(
		tcnn::cpp::create_network(color_mlp_in_dim, 3, color_mlp_network_config)
	);
	
	// Set up Optimizer
	
	json optimizer_config = {
		{"otype", "Adam"},
		{"learning_rate", 1e-2},
		{"epsilon", 1e-15},
	};
	
	optimizer = std::shared_ptr<Optimizer<tcnn::network_precision_t>>(
		create_optimizer<network_precision_t>(optimizer_config)
	);
	
	// RNG
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
}

NeRFTrainingController::~NeRFTrainingController() {
	hiprandDestroyGenerator(rng);
}

__global__ void generate_training_image_indices(
	const uint32_t n_elements,
	const uint32_t n_images,
	uint32_t* __restrict__ image_indices
) {
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx > n_elements) return;
	
	image_indices[idx] = idx / n_images;
}

__global__ void resize_floats_to_uint32_with_max(
	const uint32_t n_elements,
	const float* __restrict__ floats,
	uint32_t* __restrict__ uints,
	const float range_max
) {
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < n_elements) return;
	
	float resized_val = floats[idx] * range_max;
	uints[idx] = (uint32_t)resized_val;
}

/*__global__ void select_pixels_and_rays_from_training_data(
	const uint32_t n_elements,
	const uint32_t* __restrict__ pixel_indices,
	const uint32_t* __restrict__ image_indices,
)*/

void NeRFTrainingController::generate_next_training_batch(hipStream_t stream, uint32_t training_step, uint32_t batch_size) {
	workspace.enlarge(stream, dataset.n_pixels_per_image, dataset.images.size(), batch_size);
	
	vector<uint32_t> random_indices_host(workspace.batch_size);
	// next, pull rays from the dataset 
	hiprandGenerateUniform(rng, workspace.random_floats, workspace.batch_size);
	resize_floats_to_uint32_with_max<<<n_blocks_linear(workspace.batch_size), n_threads_linear>>>(
		workspace.batch_size, workspace.random_floats, workspace.pixel_indices, dataset.n_pixels_per_image
	);

	// need a kernel that selects ray & pixel indices from the training images


	generate_training_image_indices<<<n_blocks_linear(workspace.batch_size), n_threads_linear>>>(
		workspace.batch_size,
		dataset.images.size(),
		workspace.image_indices
	);
	// select_pixels_and_rays_from_training_data<<<n_block_linear(workspace.batch_size), n_threads_linear>>> (
	//
	//);
	
	// debug code (check indices are random)
	hipMemcpyAsync(random_indices_host.data(), workspace.image_indices, workspace.batch_size * sizeof(uint32_t), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	
	printf("%ld", random_indices_host.size());
}

void NeRFTrainingController::train_step(hipStream_t stream) {
	
	// Train the model (batch_size must be a multiple of tcnn::batch_size_granularity)
	uint32_t batch_size = tcnn::next_multiple((uint32_t)1000, tcnn::batch_size_granularity);
	
	generate_next_training_batch(stream, 1, batch_size);
	
	/*
	GPUMatrix<float> network_input(workspace.network_input);
	GPUMatrix<float> network_output(workspace.network_output);

	for (int i = 0; i < n_training_steps; ++i) {
		generate_training_batch(&training_batch_inputs, &training_batch_targets); // <-- your code

		float loss;
		model.trainer->training_step(training_stream, training_batch_inputs, training_batch_targets);
		std::cout << "iteration=" << i << " loss=" << loss << std::endl;
	}

	// Use the model
	GPUMatrix<float> inference_inputs(n_input_dims, batch_size);
	generate_inputs(&inference_inputs); // <-- your code

	GPUMatrix<float> inference_outputs(n_output_dims, batch_size);
	model.network->inference(inference_inputs, inference_outputs);
	*/
}