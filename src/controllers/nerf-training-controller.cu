#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <json/json.hpp>
#include <thrust/device_vector.h>


#include "nerf-training-controller.h"
#include "../utils/nerf-constants.cuh"
#include "../utils/occupancy-grid-kernels.cuh"
#include "../utils/training-batch-kernels.cuh"
#include "../utils/parallel-utils.cuh"

#include <iostream>
#include <fstream>

using namespace nrc;
using namespace tcnn;
using namespace nlohmann;

NeRFTrainingController::NeRFTrainingController(Dataset& dataset, NeRF* nerf)
	: dataset(dataset), nerf(nerf)
{	
	// TODO: refactor size_x to just size?
	// RNG
	// todo: CURAND_ASSERT_SUCCESS
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandGenerateSeeds(rng);

}

NeRFTrainingController::~NeRFTrainingController() {
	hiprandDestroyGenerator(rng);
}

// NeRFTrainingController member functions

void NeRFTrainingController::prepare_for_training(
	const hipStream_t& stream,
	const uint32_t& batch_size
) {
	// This allocates memory for all the elements we need during training
	workspace.enlarge(stream,
		dataset.images.size(),
		dataset.n_pixels_per_image,
		dataset.n_channels_per_image,
		batch_size,
		nerf->occupancy_grid.n_levels,
		nerf->occupancy_grid.resolution_i,
		nerf->network.get_concat_buffer_width(),
		nerf->network.get_padded_output_width()
	);

	// Create a CascadedOccupancyGrid object and copy it to the GPU
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.occ_grid, &nerf->occupancy_grid, sizeof(CascadedOccupancyGrid), hipMemcpyHostToDevice, stream)
	);

	// Copy dataset's BoundingBox to the GPU
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.bounding_box, &dataset.bounding_box, sizeof(BoundingBox), hipMemcpyHostToDevice, stream)
	);

	// Training image indices will be reused for each batch.  We select the same number of rays from each image in the dataset.
	generate_training_image_indices<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		dataset.images.size(),
		workspace.img_index
	);
	
	// Copy training cameras to the GPU
	workspace.cameras.resize_and_copy_from_host(dataset.cameras);

	// Load all images into GPU memory!
	load_images(stream);

	// Since there is no previous step here, we set the number of previous rays to the batch size
	// so that the training batch generator will generate a full batch of rays
	n_rays_in_batch = workspace.batch_size;
	training_step = 0;

	// Initialize the network
	nerf->network.prepare_for_training(stream);
}

void NeRFTrainingController::load_images(const hipStream_t& stream) {
	// make sure images are all loaded into CPU and GPU
	size_t n_image_elements = dataset.n_channels_per_image * dataset.n_pixels_per_image;
	size_t image_size = n_image_elements * sizeof(stbi_uc);
	dataset.load_images_in_parallel(
		[this, &image_size, &n_image_elements, &stream](const size_t& image_index, const TrainingImage& image) {
			CUDA_CHECK_THROW(hipMemcpyAsync(
				workspace.image_data + image_index * n_image_elements,
				image.data_cpu.get(),
				image_size,
				hipMemcpyHostToDevice,
				stream
			));
		}
	);

	printf("All images loaded to GPU.\n");
}

/**
 * Based on my understanding of the instant-ngp paper and some help from NerfAcc,
  * we must do the following to generate a batch of fixed number of samples with a dynamic number of rays
  * 
  * 0. Generate rays and pixels
  * 1. Count the number of steps each ray will take
  * 2. Determine the maximum number of rays that will fill the batch with samples
  * 3. Generate the samples (t0, t1)
  * 4. Apply stratified sampling to get an array of t-values
  * 5. Run the network forward and get the predicted color and alpha for each sample
  * 6. Accumulate the colors and alphas from the color network output, along each ray
  * 7. Calculate the loss and backpropagate
 */

void NeRFTrainingController::generate_next_training_batch(hipStream_t stream) {

	// Generate random floats for use in training

	hiprandStatus_t status = hiprandGenerateUniform(rng, workspace.random_float, workspace.batch_size);
	if (status != HIPRAND_STATUS_SUCCESS) {
		printf("Error generating random floats for training batch.\n");
	}
	
	// Convert floats to uint32_t which will be interpreted as pixel indices for any training image
	resize_floats_to_uint32_with_max<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size, workspace.random_float, workspace.pix_index, dataset.n_pixels_per_image
	);

	/**
	 * Generate rays and pixels for training
	 * 
	 * We can take a shortcut here and generate only the data needed to fill the batch back up.
	 * If not all the previous batch's rays were used, then we can reuse the unused rays.
	 * AKA, n_rays_in_batch is the number of spent rays that need to be regenerated.
	 * 
	 * Huzzah, optimization!
	 * 
	 */
	initialize_training_rays_and_pixels_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		dataset.images.size(),
		dataset.n_pixels_per_image * dataset.n_channels_per_image,
		dataset.image_dimensions,
		workspace.bounding_box,
		workspace.cameras.data(),
		workspace.image_data,
		workspace.img_index,
		workspace.pix_index,
		workspace.pix_rgba,
		workspace.ray_origin,
		workspace.ray_dir,
		workspace.ray_inv_dir,
		workspace.ray_t,
		workspace.ray_alive
	);

	CHECK_DATA(t_cpu, float, workspace.ray_t, n_rays_in_batch);

	/* Begin volumetric sampling of the previous network outputs */
	
	const float dt_min = NeRFConstants::min_step_size;
	const float dt_max = dataset.bounding_box.size_x * dt_min;
	const float cone_angle = NeRFConstants::cone_angle;

	// Count the number of steps each ray would take.  We only need to do this for the new rays.
	march_and_count_steps_per_ray_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		workspace.bounding_box,
		workspace.occ_grid,
		cone_angle,
		dt_min,
		dt_max,
		workspace.ray_dir,
		workspace.ray_inv_dir,
		workspace.ray_alive,
		workspace.ray_origin,
		workspace.ray_t,
		workspace.ray_steps
	);

	CHECK_DATA(tcpu_2, float, workspace.ray_t, n_rays_in_batch);

	/**
	 * Cumulative summation via inclusive_scan gives us the offset index that each ray's first sample should start at, relative to the start of the batch.
	 * We need to perform this cumsum over the entire batch of rays, not just the rays that were regenerated over the used ones in the previous batch.
	 */
	
	// Grab some references to the n_steps arrays
	thrust::device_ptr<uint32_t> n_steps_in_ptr(workspace.ray_steps);
	thrust::device_ptr<uint32_t> n_steps_cum_ptr(workspace.ray_steps_cum);

	// cumsum
	thrust::inclusive_scan(thrust::cuda::par.on(stream), n_steps_in_ptr, n_steps_in_ptr + workspace.batch_size, n_steps_cum_ptr);

	CHECK_DATA(nsteps_cpu, uint32_t, workspace.ray_steps, workspace.batch_size);
	CHECK_DATA(nsteps_cum_cpu, uint32_t, workspace.ray_steps_cum, workspace.batch_size);
	/**
	 * Populate the t0 and t1 buffers with the starts and ends of each ray's samples.
	 * Also copy and compact other output buffers to help with coalesced memory access in future kernels.
	 * Again, we perform this over the entire batch of samples.
	 */

	march_and_generate_samples_and_compact_buffers_kernel<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		workspace.bounding_box,
		1.0f / dataset.bounding_box.size_x,
		workspace.occ_grid,
		dt_min, dt_max,
		cone_angle,
		
		// input buffers
		workspace.ray_origin,
		workspace.ray_dir,
		workspace.ray_inv_dir,
		workspace.ray_t,
		workspace.ray_steps,
		workspace.ray_steps_cum,
		workspace.ray_alive,

		// output buffers
		workspace.network_pos,
		workspace.network_dir,
		workspace.network_dt
	);

	// Generate stratified sampling positions
	// generate_network_inputs_kernel<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
	// 	workspace.batch_size,
	// 	1.0f / dataset.bounding_box.size_x,
	// 	workspace.sample_t0,
	// 	workspace.sample_t1,
	// 	workspace.random_float,
	// 	workspace.sample_origin,
	// 	workspace.sample_dir,
	// 	workspace.network_pos,
	// 	workspace.network_dir,
	// 	workspace.network_dt
	// );

	// CHECK_DATA(dt_cpu, float, workspace.network_dt, workspace.batch_size);
	// CHECK_DATA(dir_cpu, float, workspace.network_dir, workspace.batch_size);
	// CHECK_DATA(pos_cpu, float, workspace.network_pos, workspace.batch_size);

	// Count the number of rays actually used to fill the sample batch

	int n_ray_max_idx = find_last_lt_presorted(
		stream,
		n_steps_cum_ptr,
		workspace.batch_size,
		workspace.batch_size
	);

	if (n_ray_max_idx < 0) {
		// TODO: better error handling
		throw std::runtime_error("Sample batch does not contain any rays!\n");
	}

	n_rays_in_batch = n_ray_max_idx + 1;
	hipMemcpyAsync(&n_samples_in_batch, n_steps_cum_ptr.get() + n_ray_max_idx, sizeof(uint32_t), hipMemcpyDeviceToHost, stream);
}

// just a debug tool

template <typename T>
void minmaxavg(T* arr, int n, std::string label = "") {
	// // loop through grid_dens_cpu1, calculate min, max, and average
	T min = numeric_limits<T>::max();
	T max = numeric_limits<T>::min();
	T avg = 0.0f;

	for (int i = 0; i < n; ++i) {
		T val = arr[i];
		if (val < min) {
			min = val;
		}
		if (val > max) {
			max = val;
		}
		avg += val;
	}

	avg /= n;
	printf("%s: ", label.c_str());

	printf("min: %f, max: %f, avg: %f\n", min, max, avg);
}

// update occupancy grid

void NeRFTrainingController::update_occupancy_grid(const hipStream_t& stream, const float& selection_threshold) {
	const uint32_t grid_volume = nerf->occupancy_grid.volume_i;
	const uint32_t n_bitfield_bytes = nerf->occupancy_grid.get_n_bitfield_elements();
	const uint32_t n_levels = nerf->occupancy_grid.n_levels;
	const float inv_aabb_size = 1.0f / nerf->bounding_box.size_x;
	
	// decay occupancy grid values by 0.95
	decay_occupancy_grid_values_kernel<<<n_blocks_linear(grid_volume), n_threads_linear, 0, stream>>>(
		grid_volume,
		nerf->occupancy_grid.n_levels,
		0.95f,
		nerf->occupancy_grid.get_density()
	);

	// loop through each grid level, querying the network for the density at each cell and updating the occupancy grid's density
	for (int level = 0; level < n_levels; ++level) {

		// update occupancy grid values
		uint32_t n_cells_updated = 0;
		while (n_cells_updated < grid_volume) {
			uint32_t n_cells_to_update = std::min(grid_volume - n_cells_updated, workspace.batch_size);

			uint32_t batch_size = tcnn::next_multiple(n_cells_to_update, tcnn::batch_size_granularity);
			
			// generate random floats for sampling
			hiprandStatus_t status = hiprandGenerateUniform(rng, workspace.random_float, 4 * batch_size);
			if (status != HIPRAND_STATUS_SUCCESS) {
				printf("Error generating random floats for occupancy grid update.\n");
			}

			// generate random sampling points
			generate_grid_cell_network_sample_points_kernel<<<n_blocks_linear(n_cells_to_update), n_threads_linear, 0, stream>>>(
				n_cells_to_update,
				batch_size,
				n_cells_updated,
				workspace.occ_grid,
				level,
				inv_aabb_size,
				workspace.random_float,
				workspace.network_pos
			);

			// query the density network
			nerf->network.inference(
				stream,
				batch_size,
				workspace.network_pos,
				nullptr,
				workspace.network_concat,
				workspace.network_output,
				false
			);

			CHECK_DATA(netpos_cpu, float, workspace.network_pos, n_cells_to_update);
			minmaxavg(netpos_cpu.data(), n_cells_to_update, "netpos");
			
			// update occupancy grid values
			update_occupancy_with_density_kernel<<<n_blocks_linear(n_cells_to_update), n_threads_linear, 0, stream>>>(
				n_cells_to_update,
				n_cells_updated,
				workspace.occ_grid,
				level,
				selection_threshold,
				workspace.random_float + 3 * batch_size, // (random_float + 3 * batch_size) is so thresholding doesn't correspond to x,y,z positions
				workspace.network_output + 3 * batch_size
			);

			CHECK_DATA(grid_dens_cpu, float, nerf->occupancy_grid.get_density() + grid_volume * level, grid_volume);

			minmaxavg(grid_dens_cpu.data(), grid_dens_cpu.size(), "Density Grid");
			n_cells_updated += n_cells_to_update;
		}
	}

	// update the bits by thresholding the density values

	// This is adapted from the instant-NGP paper.  See page 15 on "Updating occupancy grids"
	const float threshold = 0.01f * NeRFConstants::min_step_size;

	update_occupancy_grid_bits_kernel<<<n_blocks_linear(n_bitfield_bytes), n_threads_linear, 0, stream>>>(
		nerf->occupancy_grid.volume_i,
		n_levels,
		threshold,
		workspace.occ_grid,
		nerf->occupancy_grid.get_density(),
		nerf->occupancy_grid.get_bitfield()
	);

	CHECK_DATA(bitfield_cpu, uint8_t, nerf->occupancy_grid.get_bitfield(), n_bitfield_bytes);

	int bits_occupied = 0;
	for (int i = 0; i < n_bitfield_bytes; ++i) {
		for (int j = 0; j < n_levels; ++j) {
			if (bitfield_cpu[i] & (1 << j)) {
				++bits_occupied;
			}
		}
	}

	printf("%% of bits occupied: %f\n", 100.f * (float)bits_occupied / (grid_volume * n_levels));
}

void NeRFTrainingController::train_step(const hipStream_t& stream) {
	//printf("Training step %d...\n", training_step);

	// Generate training batch
	generate_next_training_batch(stream);

	//printf("Using %d rays and %d samples\n", n_rays_in_batch, n_samples_in_batch);
	nerf->network.train(
		stream,
		workspace.batch_size,
		n_rays_in_batch,
		n_samples_in_batch,
		workspace.ray_steps,
		workspace.ray_steps_cum,
		workspace.network_pos,
		workspace.network_dir,
		workspace.network_dt,
		workspace.pix_rgba,
		workspace.network_concat,
		workspace.network_output
	);

	++training_step;

}
