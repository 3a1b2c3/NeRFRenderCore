
#include <hip/hip_runtime.h>
#include <>
#include <json/json.hpp>
#include <Eigen/Dense>
#include <memory>
#include <thrust/device_vector.h>


#include "nerf-training-controller.h"
#include "../models/cascaded-occupancy-grid.cuh"
#include "../utils/training-batch-kernels.cuh"
#include "../utils/parallel-utils.cuh"

using namespace nrc;
using namespace Eigen;
using namespace tcnn;
using namespace nlohmann;

NeRFTrainingController::NeRFTrainingController(Dataset& dataset)
	: network(NerfNetwork(dataset.bounding_box.size_x))
{
	this->dataset = dataset;
	
	// TODO: refactor size_x to just size?
	// RNG
	// todo: CURAND_ASSERT_SUCCESS
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandGenerateSeeds(rng);

}

NeRFTrainingController::~NeRFTrainingController() {
	hiprandDestroyGenerator(rng);
}

// NeRFTrainingController member functions

void NeRFTrainingController::prepare_for_training(hipStream_t stream, uint32_t batch_size) {
	// This allocates memory for all the elements we need during training
	workspace.enlarge(stream,
		dataset.n_pixels_per_image,
		dataset.n_channels_per_image,
		dataset.images.size(),
		batch_size,
		n_occupancy_grid_levels,
		occupancy_grid_resolution
	);

	// Initialize occupancy grid bitfield (all bits set to 1)
	CUDA_CHECK_THROW(
		hipMemsetAsync(
			workspace.occupancy_grid_bitfield,
			(uint8_t)0b11111111, // set all bits to 1
			workspace.n_occupancy_grid_elements / 8,
			stream
		)
	);

	// Create a CascadedOccupancyGrid object and copy it to the GPU
	CascadedOccupancyGrid occupancy_grid_tmp(n_occupancy_grid_levels, workspace.occupancy_grid_bitfield, occupancy_grid_resolution);
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.occupancy_grid, &occupancy_grid_tmp, sizeof(CascadedOccupancyGrid), hipMemcpyHostToDevice, stream)
	);

	// Copy dataset's BoundingBox to the GPU
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.bounding_box, &dataset.bounding_box, sizeof(BoundingBox), hipMemcpyHostToDevice, stream)
	);

	// Training image indices will be reused for each batch.  We select the same number of rays from each image in the dataset.
	generate_training_image_indices<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		dataset.images.size(),
		workspace.img_index
	);
	
	// Copy training cameras to the GPU
	workspace.cameras.resize_and_copy_from_host(dataset.cameras);

	// Load all images into GPU memory!
	load_images(stream);

	// Since there is no previous step here, we set the number of previous rays to the batch size
	// so that the training batch generator will generate a full batch of rays
	n_rays_in_batch = workspace.batch_size;
	training_step = 0;

	// Initialize the network
	network.initialize_params(stream);
}

void NeRFTrainingController::load_images(hipStream_t stream) {
	// make sure images are all loaded into CPU and GPU
	size_t n_image_elements = dataset.n_channels_per_image * dataset.n_pixels_per_image;
	size_t image_size = n_image_elements * sizeof(stbi_uc);
	dataset.load_images_in_parallel(
		[this, &image_size, &n_image_elements, &stream](const size_t& image_index, const TrainingImage& image) {
			CUDA_CHECK_THROW(hipMemcpyAsync(
				workspace.image_data + image_index * n_image_elements,
				image.data_cpu.get(),
				image_size,
				hipMemcpyHostToDevice,
				stream
			));
		}
	);

	printf("All images loaded to GPU.\n");
}

/**
 * Based on my understanding of the instant-ngp paper and some help from NerfAcc,
  * we must do the following to generate a batch of fixed number of samples with a dynamic number of rays
  * 
  * 0. Generate rays and pixels
  * 1. Count the number of steps each ray will take
  * 2. Determine the maximum number of rays that will fill the batch with samples
  * 3. Generate the samples (t0, t1)
  * 4. Apply stratified sampling to get an array of t-values
  * 5. Run the network forward and get the predicted color and alpha for each sample
  * 6. Accumulate the colors and alphas from the color network output, along each ray
  * 7. Calculate the loss and backpropagate
 */

void NeRFTrainingController::generate_next_training_batch(hipStream_t stream) {

	// Generate random floats for use in training

	hiprandStatus_t status = hiprandGenerateUniform(rng, workspace.random_floats, workspace.batch_size);
	if (status != HIPRAND_STATUS_SUCCESS) {
		printf("Error generating random floats for training batch.\n");
	}
	
	// Convert floats to uint32_t which will be interpreted as pixel indices for any training image
	resize_floats_to_uint32_with_max<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size, workspace.random_floats, workspace.pix_index, dataset.n_pixels_per_image
	);

	/**
	 * Generate rays and pixels for training
	 * 
	 * We can take a shortcut here and generate only the data needed to fill the batch back up.
	 * If not all the previous batch's rays were used, then we can reuse the unused rays.
	 * AKA, n_rays_in_batch is the number of spent rays that need to be regenerated.
	 * 
	 * Huzzah, optimization!
	 * 
	 */
	initialize_training_rays_and_pixels_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		dataset.images.size(),
		dataset.n_pixels_per_image * dataset.n_channels_per_image,
		dataset.image_dimensions,
		workspace.cameras.data(),
		workspace.image_data,
		workspace.img_index,
		workspace.pix_index,
		workspace.pix_rgba,
		workspace.ray_origins,
		workspace.ray_dirs,
		workspace.ray_inv_dirs
	);

	/* Begin volumetric sampling of the previous network outputs */
	
	// TODO: calculate these accurately

	float dt_min = 0.01f;
	float dt_max = 1.0f;
	float cone_angle = 1.0f;

	// Count the number of steps each ray would take.  We only need to do this for the new rays.
	march_and_count_steps_per_ray_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		workspace.bounding_box,
		workspace.occupancy_grid,
		cone_angle,
		dt_min,
		dt_max,
		workspace.ray_origins,
		workspace.ray_dirs,
		workspace.ray_inv_dirs,
		workspace.ray_steps
	);

	/**
	 * Cumulative summation via inclusive_scan gives us the offset index that each ray's first sample should start at, relative to the start of the batch.
	 * We need to perform this cumsum over the entire batch of rays, not just the rays that were regenerated over the used ones in the previous batch.
	 */
	
	// Grab some references to the n_steps arrays
	thrust::device_ptr<uint32_t> n_steps_in_ptr(workspace.ray_steps);
	thrust::device_ptr<uint32_t> n_steps_cum_ptr(workspace.ray_steps_cumulative);

	// cumsum
	thrust::inclusive_scan(thrust::cuda::par.on(stream), n_steps_in_ptr, n_steps_in_ptr + workspace.batch_size, n_steps_cum_ptr);


	/**
	 * Populate the t0 and t1 buffers with the starts and ends of each ray's samples.
	 * Also copy and compact other output buffers to help with coalesced memory access in future kernels.
	 * Again, we perform this over the entire batch of samples.
	 */

	march_and_generate_samples_and_compact_buffers_kernel<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		workspace.bounding_box,
		workspace.occupancy_grid,
		dt_min, dt_max,
		cone_angle,
		
		// input buffers
		workspace.ray_origins,
		workspace.ray_dirs,
		workspace.ray_inv_dirs,
		workspace.ray_steps,
		workspace.ray_steps_cumulative,
		
		// output buffers
		workspace.sample_origins,
		workspace.sample_dirs,
		workspace.sample_t0,
		workspace.sample_t1
	);

	// Generate stratified sampling positions
	generate_stratified_sample_positions_kernel<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		workspace.sample_t0, workspace.sample_t1,
		workspace.random_floats,
		workspace.sample_origins,
		workspace.sample_dirs,
		workspace.sample_positions,
		workspace.sample_dt
	);

	// Count the number of rays actually used to fill the sample batch

	int n_ray_max_idx = find_last_lt_presorted(
		stream,
		n_steps_cum_ptr,
		workspace.batch_size,
		workspace.batch_size
	);

	if (n_ray_max_idx <= 0) {
		// TODO: better error handling
		throw std::runtime_error("Sample batch does not contain any rays!\n");
	}

	n_rays_in_batch = n_ray_max_idx + 1;
	hipMemcpyAsync(&n_samples_in_batch, n_steps_cum_ptr.get() + n_ray_max_idx, sizeof(uint32_t), hipMemcpyDeviceToHost, stream);
}

void NeRFTrainingController::train_step(hipStream_t stream) {
	printf("Training step %d...\n", training_step);

	// Generate training batch
	generate_next_training_batch(stream);

	printf("Using %d rays and %d samples\n", n_rays_in_batch, n_samples_in_batch);
	
	// TODO: NORMALIZE SAMPLE_DIRS AND SAMPLE_POSITIONS (THANK YOU @BURIEDANIMAL)
	network.train_step(
		stream,
		workspace.batch_size,
		n_rays_in_batch,
		n_samples_in_batch,
		workspace.ray_steps,
		workspace.ray_steps_cumulative,
		workspace.sample_positions,
		workspace.sample_dirs,
		workspace.sample_dt,
		workspace.pix_rgba
	);

	++training_step;

}
