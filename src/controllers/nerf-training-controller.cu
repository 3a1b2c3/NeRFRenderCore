#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <json/json.hpp>
#include <thrust/device_vector.h>
#include <tiny-cuda-nn/common.h>


#include "nerf-training-controller.h"

#include "../utils/stream-compaction.cuh"
#include "../utils/nerf-constants.cuh"
#include "../utils/occupancy-grid-kernels.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/training-batch-kernels.cuh"

#include "../common.h"

#include <iostream>
#include <fstream>

using namespace nrc;
using namespace tcnn;
using namespace nlohmann;

NeRFTrainingController::NeRFTrainingController(Dataset& dataset, NeRF* nerf)
	: dataset(dataset), nerf(nerf)
{	
	// TODO: refactor size_x to just size?
	// RNG
	// todo: CURAND_ASSERT_SUCCESS
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandGenerateSeeds(rng);

}

NeRFTrainingController::~NeRFTrainingController() {
	hiprandDestroyGenerator(rng);
}

// NeRFTrainingController member functions

void NeRFTrainingController::prepare_for_training(
	const hipStream_t& stream,
	const uint32_t& batch_size
) {
	// This allocates memory for all the elements we need during training
	workspace.enlarge(stream,
		dataset.images.size(),
		dataset.n_pixels_per_image,
		dataset.n_channels_per_image,
		batch_size,
		nerf->occupancy_grid.n_levels,
		nerf->occupancy_grid.resolution_i,
		nerf->network.get_concat_buffer_width(),
		nerf->network.get_padded_output_width()
	);

	// Create a CascadedOccupancyGrid object and copy it to the GPU
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.occ_grid, &nerf->occupancy_grid, sizeof(CascadedOccupancyGrid), hipMemcpyHostToDevice, stream)
	);

	// Copy dataset's BoundingBox to the GPU
	CUDA_CHECK_THROW(
		hipMemcpyAsync(workspace.bounding_box, &dataset.bounding_box, sizeof(BoundingBox), hipMemcpyHostToDevice, stream)
	);

	// Training image indices will be reused for each batch.  We select the same number of rays from each image in the dataset.
	generate_training_image_indices<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size,
		dataset.images.size(),
		workspace.img_index
	);
	
	// Copy training cameras to the GPU
	workspace.cameras.resize_and_copy_from_host(dataset.cameras);

	// Load all images into GPU memory!
	load_images(stream);

	// Since there is no previous step here, we set the number of previous rays to the batch size
	// so that the training batch generator will generate a full batch of rays
	n_rays_in_batch = workspace.batch_size;
	training_step = 0;

	// Initialize the network
	nerf->network.prepare_for_training(stream);
}

void NeRFTrainingController::load_images(const hipStream_t& stream) {
	// make sure images are all loaded into CPU and GPU
	size_t n_image_elements = dataset.n_channels_per_image * dataset.n_pixels_per_image;
	size_t image_size = n_image_elements * sizeof(stbi_uc);

	dataset.load_images_in_parallel(
		[this, &image_size, &n_image_elements, &stream](const size_t& image_index, const TrainingImage& image) {
			CUDA_CHECK_THROW(hipMemcpyAsync(
				workspace.image_data + image_index * n_image_elements,
				image.data_cpu.get(),
				image_size,
				hipMemcpyHostToDevice,
				stream
			));
		}
	);

	printf("All images loaded to GPU.\n");
}

/**
 * generate_next_training_batch does the following:
  * 
  * 0. Generate rays and ground truth pixels from training cameras and images
  * 1. Count the number of steps each ray will take
  * 2. Determine the maximum number of rays that will fill the batch with samples
  * 3. Generate the samples for the density network
  * 4. Query the network, determine:
  * 	4a. Which samples are visible?
  * 	4b. What is the transmittance of each sample?
  * 5. Compact this batch of samples, excluding the invisible ones
  * 6. Repeat steps 0-5 until the batch is full enough
 */

void NeRFTrainingController::generate_next_training_batch(hipStream_t stream) {

	// Generate random floats for use in training

	hiprandStatus_t status = hiprandGenerateUniform(rng, workspace.random_float, workspace.batch_size);
	if (status != HIPRAND_STATUS_SUCCESS) {
		printf("Error generating random floats for training batch.\n");
	}
	
	// Convert floats to uint32_t which will be interpreted as pixel indices for any training image
	resize_floats_to_uint32_with_max<<<n_blocks_linear(workspace.batch_size), n_threads_linear, 0, stream>>>(
		workspace.batch_size, workspace.random_float, workspace.pix_index, dataset.n_pixels_per_image
	);

	/**
	 * Generate rays and pixels for training
	 * 
	 * We can take a shortcut here and generate only the data needed to fill the batch back up.
	 * If not all the previous batch's rays were used, then we only need to regenerate rays
	 * for batch_size minus the number of rays that were used.
	 */

	initialize_training_rays_and_pixels_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		dataset.images.size(),
		dataset.n_pixels_per_image * dataset.n_channels_per_image,
		dataset.image_dimensions,
		workspace.bounding_box,
		workspace.cameras.data(),
		workspace.image_data,
		workspace.img_index,
		workspace.pix_index,
		workspace.pix_rgba[0],
		workspace.ray_origin,
		workspace.ray_dir[0],
		workspace.ray_inv_dir,
		workspace.ray_t,
		workspace.ray_alive
	);

	// Count the number of steps each ray will take
	
	const float dt_min = NeRFConstants::min_step_size;
	const float dt_max = dataset.bounding_box.size_x * dt_min;
	const float cone_angle = NeRFConstants::cone_angle;

	// Count the number of steps each ray would take.  We only need to do this for the new rays.
	march_and_count_steps_per_ray_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		workspace.bounding_box,
		workspace.occ_grid,
		cone_angle,
		dt_min,
		dt_max,
		workspace.ray_dir[0],
		workspace.ray_inv_dir,
		workspace.ray_alive,
		workspace.ray_origin,
		workspace.ray_t,
		workspace.ray_step[0]
	);

	CHECK_DATA(ray_dir_cpu, float, workspace.ray_dir[0], n_rays_in_batch * 3);

	// Count the number of rays that will fill the batch with the maximum number of samples
	/**
	 * Cumulative summation via inclusive_scan gives us the offset index that each ray's first sample should start at, relative to the start of the batch.
	 * We need to perform this cumsum over the entire batch of rays, not just the rays that were regenerated over the used ones in the previous batch.
	 */

	// Grab some references to the n_steps arrays
	thrust::device_ptr<uint32_t> n_steps_ptr(workspace.ray_step[0]);
	thrust::device_ptr<uint32_t> ray_offset_ptr(workspace.ray_offset[0]);
	
	// cumulative sum the number of steps for each ray
	thrust::exclusive_scan(
		thrust::cuda::par.on(stream),
		n_steps_ptr,
		n_steps_ptr + workspace.batch_size,
		ray_offset_ptr
	);

	// Count the number of rays actually used to fill the sample batch
	const int n_ray_max_idx = find_last_lt_presorted(
		stream,
		ray_offset_ptr,
		workspace.batch_size,
		workspace.batch_size
	) - 1;

	if (n_rays_in_batch < 0) {
		throw std::runtime_error("No rays were generated for this training batch!\n");
	}

	n_rays_in_batch = static_cast<uint32_t>(n_ray_max_idx + 1);

	// Count the number of samples that will be generated
	CUDA_CHECK_THROW(
		hipMemcpyAsync(
			&n_samples_in_batch,
			ray_offset_ptr.get() + n_rays_in_batch,
			sizeof(uint32_t),
			hipMemcpyDeviceToHost,
			stream
		)
	);

	CUDA_CHECK_THROW(hipStreamSynchronize(stream));

	if (n_samples_in_batch < 1) {
		throw std::runtime_error("No samples were generated for this training batch!\n");
	}
	
	// Generate sample positions
	march_and_generate_network_positions_kernel<<<n_blocks_linear(n_rays_in_batch), n_threads_linear, 0, stream>>>(
		n_rays_in_batch,
		workspace.batch_size,
		workspace.bounding_box,
		1.0f / dataset.bounding_box.size_x,
		workspace.occ_grid,
		dt_min,
		dt_max,
		cone_angle,

		// input buffers
		workspace.ray_origin,
		workspace.ray_dir[0],
		workspace.ray_inv_dir,
		workspace.ray_t,
		workspace.ray_offset[0],
		workspace.ray_alive,

		// dual-use buffers
		workspace.ray_step[0],

		// output buffers
		workspace.sample_pos[0],
		workspace.sample_dir,
		workspace.sample_dt[0]
	);
}

// update occupancy grid

void NeRFTrainingController::update_occupancy_grid(const hipStream_t& stream, const float& selection_threshold) {
	const uint32_t grid_volume = nerf->occupancy_grid.volume_i;
	const uint32_t n_bitfield_bytes = nerf->occupancy_grid.get_n_bitfield_elements();
	const uint32_t n_levels = nerf->occupancy_grid.n_levels;
	const float inv_aabb_size = 1.0f / nerf->bounding_box.size_x;
	
	// decay occupancy grid values by 0.95
	decay_occupancy_grid_values_kernel<<<n_blocks_linear(grid_volume), n_threads_linear, 0, stream>>>(
		grid_volume,
		nerf->occupancy_grid.n_levels,
		0.95f,
		nerf->occupancy_grid.get_density()
	);

	// loop through each grid level, querying the network for the density at each cell and updating the occupancy grid's density
	for (int level = 0; level < n_levels; ++level) {

		// update occupancy grid values
		uint32_t n_cells_updated = 0;
		while (n_cells_updated < grid_volume) {
			uint32_t n_cells_to_update = std::min(grid_volume - n_cells_updated, workspace.batch_size);

			uint32_t batch_size = tcnn::next_multiple(n_cells_to_update, tcnn::batch_size_granularity);
			
			// generate random floats for sampling
			hiprandStatus_t status = hiprandGenerateUniform(rng, workspace.random_float, 4 * batch_size);
			if (status != HIPRAND_STATUS_SUCCESS) {
				printf("Error generating random floats for occupancy grid update.\n");
			}

			// generate random sampling points
			generate_grid_cell_network_sample_points_kernel<<<n_blocks_linear(n_cells_to_update), n_threads_linear, 0, stream>>>(
				n_cells_to_update,
				batch_size,
				n_cells_updated,
				workspace.occ_grid,
				level,
				inv_aabb_size,
				workspace.random_float,
				workspace.sample_pos[0]
			);

			// query the density network
			nerf->network.inference(
				stream,
				batch_size,
				workspace.sample_pos[0],
				nullptr,
				workspace.network_concat,
				workspace.network_output,
				false
			);

			// update occupancy grid values
			update_occupancy_with_density_kernel<<<n_blocks_linear(n_cells_to_update), n_threads_linear, 0, stream>>>(
				n_cells_to_update,
				n_cells_updated,
				level,
				selection_threshold,
				workspace.random_float + 3 * batch_size, // (random_float + 3 * batch_size) is so thresholding doesn't correspond to x,y,z positions
				workspace.network_output + 3 * batch_size,
				workspace.occ_grid
			);

			n_cells_updated += n_cells_to_update;
		}
	}

	// update the bits by thresholding the density values

	// This is adapted from the instant-NGP paper.  See page 15 on "Updating occupancy grids"
	// For some reason, the way the paper says it does not work for this implementation.
	// It seems to work with a threshold of 0.01
	const float threshold = 0.01f;// * NeRFConstants::min_step_size;

	update_occupancy_grid_bits_kernel<<<n_blocks_linear(n_bitfield_bytes), n_threads_linear, 0, stream>>>(
		nerf->occupancy_grid.volume_i,
		n_levels,
		threshold,
		workspace.occ_grid,
		nerf->occupancy_grid.get_density(),
		nerf->occupancy_grid.get_bitfield()
	);

	CHECK_DATA(bitfield_cpu, uint8_t, nerf->occupancy_grid.get_bitfield(), n_bitfield_bytes);

	// this is just debug code to print out the percentage of bits occupied
	int bits_occupied = 0;
	for (int i = 0; i < n_bitfield_bytes; ++i) {
		for (int j = 0; j < n_levels; ++j) {
			if (bitfield_cpu[i] & (1 << j)) {
				++bits_occupied;
			}
		}
	}

	printf("%% of bits occupied: %f\n", 100.f * (float)bits_occupied / (grid_volume * n_levels));
}

void NeRFTrainingController::train_step(const hipStream_t& stream) {
	// Generate training batch
	generate_next_training_batch(stream);
	nerf->network.train(
		stream,
		workspace.batch_size,
		n_rays_in_batch,
		n_samples_in_batch,
		workspace.ray_step[0],
		workspace.ray_offset[0],
		workspace.sample_pos[0],
		workspace.sample_dir,
		workspace.sample_dt[0],
		workspace.pix_rgba[0],
		workspace.network_concat,
		workspace.network_output
	);

	++training_step;

}
