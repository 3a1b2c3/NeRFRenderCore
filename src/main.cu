﻿#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <json/json.hpp>

#include "common.h"
#include "main.h"
#include "models/dataset.h"
#include "models/cascaded-occupancy-grid.cuh"
#include "controllers/nerf-training-controller.h"

int main()
{
	nrc::Dataset dataset = nrc::Dataset("E:\\2022\\nerf-library\\testdata\\lego\\transforms.json");
    
    // nrc::OccupancyGrid grid(1);

    // printf("%lu", grid.max_index());
    auto controller = nrc::NeRFTrainingController(dataset);
    
    hipStream_t stream;
	CUDA_CHECK_THROW(hipStreamCreate(&stream));

    controller.prepare_for_training(stream, 2<<21);
    
    for (int i = 0; i < 1000; ++i) {
        controller.train_step(stream);
    }

    // Wait for the kernel to finish executing
    hipDeviceSynchronize();

    return 0;
}
