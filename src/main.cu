#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <json/json.hpp>

#include "common.h"
#include "main.h"
#include "models/camera.cuh"
#include "models/dataset.h"
#include "models/cascaded-occupancy-grid.cuh"
#include "models/render-buffer.cuh"
#include "models/render-request.cuh"
#include "controllers/nerf-training-controller.h"
#include "controllers/nerf-rendering-controller.h"
#include "services/nerf-manager.cuh"
#include "utils/linalg.cuh"

#include "utils/alphatensor_mmul4x4.cuh"
#include "utils/coordinate-transformations.cuh"
#include "utils/linalg/transform4f.cuh"

#include <tiny-cuda-nn/common.h>

#include "models/cascaded-occupancy-grid.cuh"
int main()
{

	float ray_ori_x = 1.0f;
	float ray_ori_y = 1.0f;
	float ray_ori_z = 1.0f;
	float ray_dir_x = 1.0f / sqrt(3.0f);
	float ray_dir_y = 1.0f / sqrt(3.0f);
	float ray_dir_z = 1.0f / sqrt(3.0f);
	float ray_idir_x = 1.0f / ray_dir_x;
	float ray_idir_y = 1.0f / ray_dir_y;
	float ray_idir_z = 1.0f / ray_dir_z;

	nrc::CascadedOccupancyGrid grid(5);

	const float dt = grid.get_dt_to_next_voxel(
		ray_ori_x, ray_ori_y, ray_ori_z,
		ray_dir_x, ray_dir_y, ray_dir_z,
		ray_idir_x, ray_idir_y, ray_idir_z,
		0.01 * sqrt(3.0f) / 1024.0f,
		4
	);

	nrc::Dataset dataset = nrc::Dataset("E:\\2022\\nerf-library\\testdata\\lego\\transforms.json");
	// auto dataset = nrc::Dataset("E:\\2022\\nerf-library\\FascinatedByFungi2022\\big-white-chanterelle\\transforms.json");
	auto nerf_manager = nrc::NeRFManager();
	
	// nrc::OccupancyGrid grid(1);

	// printf("%lu", grid.max_index());
	
	hipStream_t stream;
	CUDA_CHECK_THROW(hipStreamCreate(&stream));

	auto nerf = nerf_manager.create_trainable_nerf(stream, dataset.bounding_box);

	// set up training controller
	auto trainer = nrc::NeRFTrainingController(dataset, nerf);
	trainer.prepare_for_training(stream, 1<<21);

	// set up rendering controller
	auto renderer = nrc::NeRFRenderingController();
	float* rgba;

	CUDA_CHECK_THROW(hipMallocManaged(&rgba, 1024 * 1024 * 4 * sizeof(float)));
	auto render_buffer = nrc::RenderBuffer(1024, 1024, rgba);

	auto camera_transform = nrc::Matrix4f::Identity();
	auto cam6 = dataset.cameras[6];
	auto cam0 = dataset.cameras[6];

	// fetch nerfs as pointers
	std::vector<nrc::NeRF*> nerf_ptrs;
	for (auto& nerf : nerf_manager.get_nerfs()) {
		nerf_ptrs.emplace_back(nerf);
	}

	for (int i = 0; i <= 100000; ++i) {
		trainer.train_step(stream);

		// every 16 training steps, update the occupancy grid

		if (i % 16 == 0 && i > 0) {
			// only threshold to 50% after 256 training steps, otherwise select 100% of the cells
			const float cell_selection_threshold = i > 256 ? 0.5f : 1.0f;
			trainer.update_occupancy_grid(stream, cell_selection_threshold);
		}

		if (i % 1000 == 0 && i > 0) {
			float progress = 0.0f;//(float)i / (30.0f * 60.0f);
			float tau = 2.0f * 3.14159f;
			auto tform = nrc::Matrix4f::Rotation(3.0f * progress * tau, 0.0f, 1.0f, 0.0f) * cam0.transform;
			auto render_cam = nrc::Camera(
				cam0.near,
				cam0.far,
				cam0.focal_length,
				make_int2(1024, 1024),
				cam0.sensor_size,
				tform
			);

			auto render_request = nrc::RenderRequest(render_buffer, render_cam, nerf_ptrs);
			render_request.output.clear(stream);
			renderer.request_render(stream, render_request);
			render_request.output.save_image(stream, fmt::format("H:\\test-render-2\\step-{}.png", i));
		}
	}

	// Wait for the kernel to finish executing
	hipDeviceSynchronize();
	return 0;
}
