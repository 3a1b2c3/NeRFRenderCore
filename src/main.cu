#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <json/json.hpp>

#include "common.h"
#include "main.h"
#include "models/camera.cuh"
#include "models/dataset.h"
#include "models/cascaded-occupancy-grid.cuh"
#include "models/render-buffer.cuh"
#include "models/render-request.cuh"
#include "controllers/nerf-training-controller.h"
#include "controllers/nerf-rendering-controller.h"
#include "services/nerf-manager.cuh"
#include "utils/linalg.cuh"

#include "utils/coordinate-transformations.cuh"
#include "utils/linalg/transform4f.cuh"

#include <tiny-cuda-nn/common.h>

#include "models/cascaded-occupancy-grid.cuh"
int main()
{

	auto og = nrc::CascadedOccupancyGrid(1, 16);

	auto x = og.get_voxel_morton_index(2, 0.5f, -0.5f, 0.5f);
	float ix, iy, iz;
	og.get_voxel_xyz_from_morton_index(x, 2, ix, iy, iz);

	nrc::Dataset dataset = nrc::Dataset("E:\\2022\\nerf-library\\testdata\\lego\\transforms.json");
	// auto dataset = nrc::Dataset("E:\\2022\\nerf-library\\FascinatedByFungi2022\\big-white-chanterelle\\transforms.json");
	auto nerf_manager = nrc::NeRFManager();
	
	// nrc::OccupancyGrid grid(1);

	// printf("%lu", grid.max_index());
	
	hipStream_t stream;
	CUDA_CHECK_THROW(hipStreamCreate(&stream));

	auto nerf = nerf_manager.create_trainable_nerf(stream, dataset.bounding_box);

	// set up training controller
	auto trainer = nrc::NeRFTrainingController(dataset, nerf);
	trainer.prepare_for_training(stream, 2<<20);

	// set up rendering controller
	auto renderer = nrc::NeRFRenderingController();
	float* rgba;

	CUDA_CHECK_THROW(hipMallocManaged(&rgba, 1024 * 1024 * 4 * sizeof(float)));
	auto render_buffer = nrc::RenderBuffer(1024, 1024, rgba);

	auto camera_transform = nrc::Matrix4f::Identity();
	auto cam6 = dataset.cameras[6];
	auto cam0 = dataset.cameras[6];

	// fetch nerfs as pointers
	std::vector<nrc::NeRF*> nerf_ptrs;
	for (auto& nerf : nerf_manager.get_nerfs()) {
		nerf_ptrs.emplace_back(nerf);
	}

	for (int i = 0; i <= 100000; ++i) {
		trainer.train_step(stream);
		// every 16 training steps, update the occupancy grid

		if (i % 16 == 0 && i > 16) {
			// only threshold to 50% after 256 training steps, otherwise select 100% of the cells
			const float cell_selection_threshold = i > 256 ? 0.5f : 1.0f;
			trainer.update_occupancy_grid(stream, cell_selection_threshold);
		}

		if (i > 128 && i % 128 == 0) {
			float progress = (float)i / 360.0f;
			float tau = 2.0f * 3.14159f;
			auto tform = nrc::Matrix4f::Rotation(progress * tau, 0.0f, 1.0f, 0.0f) * cam0.transform;
			auto render_cam = nrc::Camera(
				cam0.near,
				cam0.far,
				cam0.focal_length,
				make_int2(1024, 1024),
				cam0.sensor_size,
				tform
			);

			auto render_request = nrc::RenderRequest(render_buffer, render_cam, nerf_ptrs);
			render_request.output.clear(stream);
			renderer.request_render(stream, render_request);
			render_request.output.save_image(stream, fmt::format("H:\\good\\step-{}.png", i));
		}
	}

	// Wait for the kernel to finish executing
	hipDeviceSynchronize();
	return 0;
}
